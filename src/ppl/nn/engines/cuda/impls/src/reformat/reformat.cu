#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include <float.h>

#include "cudakernel/reformat/reformat.h"
#include "cudakernel/common/common.h"
#include "cudakernel/common/divmod_fast.h"
#include "cudakernel/common/macro.h"

#include "hip/hip_fp16.h"
using namespace PPLCUDA;
using namespace ppl::nn;
using namespace ppl::common;

#define DIM 32
#define LEASTCHANNEL 16
template <typename T, CVTFormatMode mode>
__global__ void cuda_kernel_cvtformat(
    T* input,
    T* output,
    ReFormatParam param)
{
}

#define cvtNCTONHWC8(type)                                                                               \
template<>                                                                                              \
__global__ void cuda_kernel_cvtformat<type, NDARRAY_NHWC8>(                                              \
    type* input,                                                                                        \
    type* output,                                                                                       \
    ReFormatParam param)                                                                                \
{                                                                                                       \
    __shared__ type share_val[DIM][DIM + 1];                                                            \
                                                                                                        \
    int64_t num = blockIdx.z;                                                                           \
    for (int n = num; n < param.n_outer; n+= blockDim.x) {                                              \
        int64_t idx_w = blockIdx.x * blockDim.x + threadIdx.x;                                          \
        int64_t idx_h = blockIdx.y * blockDim.y + threadIdx.y;                                          \
                                                                                                        \
        if (idx_w < param.n_inner && idx_h < param.src_pad) {                                           \
            int64_t offset = n * param.src_pad * param.n_inner + idx_h * param.n_inner + idx_w;         \
            share_val[threadIdx.y][threadIdx.x] = input[offset];                                        \
        } else {                                                                                        \
            share_val[threadIdx.y][threadIdx.x] = (type)0;                                              \
        }                                                                                               \
        __syncthreads();                                                                                \
                                                                                                        \
        idx_w = blockIdx.y * blockDim.y + threadIdx.x;                                                  \
        idx_h = blockIdx.x * blockDim.x + threadIdx.y;                                                  \
                                                                                                        \
        if (idx_w < param.dst_pad && idx_h < param.n_inner) {                                           \
            int64_t offset = n * param.dst_pad * param.n_inner + idx_h * param.dst_pad + idx_w;         \
            output[offset] = share_val[threadIdx.x][threadIdx.y];                                       \
        }                                                                                               \
    }                                                                                                   \
}

#if __CUDACC_VER_MAJOR__ >= 9
    cvtNCTONHWC8(half)
#endif
    cvtNCTONHWC8(float)
    cvtNCTONHWC8(char)
    cvtNCTONHWC8(double)



#define cvtNHWC8TONC(type)                                                                               \
template<>                                                                                              \
__global__ void cuda_kernel_cvtformat<type, NHWC8_NDARRAY>(                                              \
    type* input,                                                                                        \
    type* output,                                                                                       \
    ReFormatParam param)                                                                                \
{                                                                                                       \
    __shared__ type share_val[DIM][DIM + 1];                                                            \
                                                                                                        \
    int64_t num = blockIdx.z;                                                                           \
    for (int n = num; n < param.n_outer; n += blockDim.x) {                                              \
        int64_t idx_w = blockIdx.x * blockDim.x + threadIdx.x;                                          \
        int64_t idx_h = blockIdx.y * blockDim.y + threadIdx.y;                                          \
                                                                                                        \
        if (idx_w < param.src_pad && idx_h < param.n_inner) {                                           \
            int64_t offset = n * param.src_pad * param.n_inner + idx_h * param.src_pad + idx_w;         \
            share_val[threadIdx.y][threadIdx.x] = input[offset];                                        \
        } else {                                                                                        \
            share_val[threadIdx.y][threadIdx.x] = (type)0;                                              \
        }                                                                                               \
        __syncthreads();                                                                                \
                                                                                                        \
        idx_w = blockIdx.y * blockDim.y + threadIdx.x;                                                  \
        idx_h = blockIdx.x * blockDim.x + threadIdx.y;                                                  \
                                                                                                        \
        if (idx_w < param.n_inner && idx_h < param.dst_pad) {                                           \
            int64_t offset = n * param.dst_pad * param.n_inner + idx_h * param.n_inner + idx_w;         \
            output[offset] = share_val[threadIdx.x][threadIdx.y];                                       \
        }                                                                                               \
    }                                                                                                   \
}

#if __CUDACC_VER_MAJOR__ >= 9
    cvtNHWC8TONC(half)
#endif
    cvtNHWC8TONC(float)
    cvtNHWC8TONC(char)
    cvtNHWC8TONC(double)



#define cvtN4CXTONC(type)                                                                                              \
template <>                                                                                                            \
__global__ void cuda_kernel_cvtformat<type, N4CX_NDARRAY>(                                                             \
    type * input,                                                                                                      \
    type * output,                                                                                                     \
    ReFormatParam param)                                                                                               \
{                                                                                                                      \
    const uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;                                                        \
    if (tid >= param.n_inner)                                                                                          \
        return;                                                                                                        \
    const uint64_t inner_idx = tid;                                                                                    \
    const uint64_t num_inner = blockIdx.z;                                                                             \
    const uint64_t c4_idx    = blockIdx.y;                                                                             \
    _Pragma("unroll 4") for (int c_in_c4_idx = 0; c_in_c4_idx < 4; c_in_c4_idx++)                                      \
    {                                                                                                                  \
        const uint64_t c_idx       = c4_idx * 4 + c_in_c4_idx;                                                         \
        const uint64_t size        = param.n_inner;                                                                    \
        const uint64_t padChannels = gridDim.y * 4;                                                                    \
        const uint64_t numChannels = param.channel;                                                                    \
        if (c_idx < numChannels) {                                                                                     \
            const uint64_t offset    = num_inner * padChannels * size + (c4_idx * size + inner_idx) * 4 + c_in_c4_idx; \
            const uint64_t outOffset = num_inner * numChannels * size + c_idx * size + inner_idx;                      \
            output[outOffset]        = input[offset];                                                                  \
        }                                                                                                              \
    }                                                                                                                  \
}

#if __CUDACC_VER_MAJOR__ >= 9
    cvtN4CXTONC(half)
#endif
    cvtN4CXTONC(float)
    cvtN4CXTONC(char)
    cvtN4CXTONC(double)



#define cvtNCTON4CX(type)                                                                                             \
template <>                                                                                                           \
__global__ void cuda_kernel_cvtformat<type, NDARRAY_N4CX>(                                                            \
    type * input,                                                                                                     \
    type * output,                                                                                                    \
    ReFormatParam param)                                                                                              \
{                                                                                                                     \
    const uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;                                                       \
    if (tid >= param.n_inner)                                                                                         \
        return;                                                                                                       \
    const uint64_t inner_idx = tid;                                                                                   \
    const uint64_t num_inner = blockIdx.z;                                                                            \
    const uint64_t c4_idx    = blockIdx.y;                                                                            \
    _Pragma("unroll 4") for (int c_in_c4_idx = 0; c_in_c4_idx < 4; c_in_c4_idx++)                                     \
    {                                                                                                                 \
        const uint64_t c_idx       = c4_idx * 4 + c_in_c4_idx;                                                        \
        const uint64_t size        = param.n_inner;                                                                   \
        const uint64_t padChannels = gridDim.y * 4;                                                                   \
        const uint64_t numChannels = param.channel;                                                                   \
        if (c_idx < numChannels) {                                                                                    \
            const uint64_t offset   = num_inner * padChannels * size + (c4_idx * size + inner_idx) * 4 + c_in_c4_idx; \
            const uint64_t inOffset = num_inner * numChannels * size + c_idx * size + inner_idx;                      \
            output[offset]          = input[inOffset];                                                                \
        }                                                                                                             \
    }                                                                                                                 \
}

#if __CUDACC_VER_MAJOR__ >= 9
    cvtNCTON4CX(half)
#endif
    cvtNCTON4CX(float)
    cvtNCTON4CX(char)
    cvtNCTON4CX(double)



template <typename T, CVTFormatMode mode>
__global__ void cuda_kernel_small_channel_cvtformat(
    T* input,
    int num_elems,
    DivModFast inner_fast,
    DivModFast src_pad_fast,
    DivModFast dst_pad_fast,
    T* output,
    ReFormatParam param)
{
}
/*
// #define cvtSMCHANNELNCTONHWC8(type)                                                                      \
// template<>                                                                                              \
// __global__ void cuda_kernel_small_channel_cvtformat<type, NDARRAY_NHWC8>(                                \
//     type* input,                                                                                        \
//     int64_t num_elems,                                                                                  \
//     type* output,                                                                                       \
//     ReFormatParam param)                                                                                \
// {                                                                                                       \
//     int64_t tid = blockIdx.x * blockDim.x + threadIdx.x;                                                \
//     if (tid >= num_elems) return;                                                                       \
//     int c_idx = tid % param.dst_pad;                                                                    \
//     int inner_idx = (tid / param.dst_pad) % param.n_inner;                                              \
//     int outer_idx = tid / (param.dst_pad * param.n_inner);                                              \
//     int64_t offset = outer_idx * param.src_pad * param.n_inner + c_idx * param.n_inner + inner_idx;     \
//     output[tid] = c_idx > param.channel ? input[offset] : (type)0;                                      \
// }
*/
#define cvtSMCHANNELNCTONHWC8(type)                                                                      \
template<>                                                                                              \
__global__ void cuda_kernel_small_channel_cvtformat<type, NDARRAY_NHWC8>(                                \
    type* input,                                                                                        \
    int num_elems,                                                                                      \
    DivModFast inner_fast,                                                                              \
    DivModFast src_pad_fast,                                                                            \
    DivModFast dst_pad_fast,                                                                            \
    type* output,                                                                                       \
    ReFormatParam param)                                                                                \
{                                                                                                       \
    int tid = blockIdx.x * blockDim.x + threadIdx.x;                                                    \
    if (tid >= num_elems) return;                                                                       \
    int inner_idx = 0, num_inner = 0, c_idx = 0;                                                        \
    dst_pad_fast.divmod(tid, num_inner, c_idx);                                                         \
    inner_idx = inner_fast.mod(num_inner);                                                              \
    int outer_idx = inner_fast.div(num_inner);                                                              \
    int offset = outer_idx * param.src_pad * param.n_inner + c_idx * param.n_inner + inner_idx;         \
    output[tid] =  c_idx < param.src_pad ? input[offset] : (type)0;                                     \
}

#if __CUDACC_VER_MAJOR__ >= 9
    cvtSMCHANNELNCTONHWC8(half)
#endif
    cvtSMCHANNELNCTONHWC8(float)
    cvtSMCHANNELNCTONHWC8(char)
    cvtSMCHANNELNCTONHWC8(double)



#define cvtSMCHANNELNHWC8TONC(type)                                                                      \
template<>                                                                                              \
__global__ void cuda_kernel_small_channel_cvtformat<type, NHWC8_NDARRAY>(                                \
    type* input,                                                                                        \
    int num_elems,                                                                                      \
    DivModFast inner_fast,                                                                              \
    DivModFast src_pad_fast,                                                                            \
    DivModFast dst_pad_fast,                                                                            \
    type* output,                                                                                       \
    ReFormatParam param)                                                                                \
{                                                                                                       \
    int tid = blockIdx.x * blockDim.x + threadIdx.x;                                                    \
    if (tid >= num_elems) return;                                                                       \
    int inner_idx = 0, num_inner = 0, c_idx = 0;                                                        \
    inner_fast.divmod(tid, num_inner, inner_idx);                                                       \
    c_idx = dst_pad_fast.mod(num_inner);                                                                \
    int outer_idx = tid / (param.dst_pad * param.n_inner);                                              \
    int offset = outer_idx * param.src_pad * param.n_inner + c_idx + inner_idx * param.src_pad;         \
    output[tid] = input[offset];                                                                        \
}

#if __CUDACC_VER_MAJOR__ >= 9
    cvtSMCHANNELNHWC8TONC(half)
#endif
    cvtSMCHANNELNHWC8TONC(float)
    cvtSMCHANNELNHWC8TONC(char)
    cvtSMCHANNELNHWC8TONC(double)



#define cvtSMCHANNELN4CXTONC(type)                                                                               \
template <>                                                                                                      \
__global__ void cuda_kernel_small_channel_cvtformat<type, N4CX_NDARRAY>(                                         \
    type * input,                                                                                                \
    int num_elems,                                                                                               \
    DivModFast inner_fast,                                                                                       \
    DivModFast src_pad_fast,                                                                                     \
    DivModFast dst_pad_fast,                                                                                     \
    type* output,                                                                                                \
    ReFormatParam param)                                                                                         \
{                                                                                                                \
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;                                                       \
    if (tid >= num_elems)                                                                                        \
        return;                                                                                                  \
    int inner_idx, num_inner, c_idx;                                                                             \
    inner_fast.divmod(tid, num_inner, inner_idx);                                                                \
    src_pad_fast.divmod(num_inner, num_inner, c_idx);                                                            \
    const int c4_idx           = c_idx / 4;                                                                      \
    const int c_in_c4_idx      = c_idx % 4;                                                                      \
    const uint64_t size        = param.n_inner;                                                                  \
    const uint64_t padChannels = param.src_pad;                                                                  \
    const uint64_t numChannels = param.channel;                                                                  \
    const uint64_t offset      = num_inner * padChannels * size + (c4_idx * size + inner_idx) * 4 + c_in_c4_idx; \
    const uint64_t outOffset   = num_inner * numChannels * size + c_idx * size + inner_idx;                      \
    output[outOffset]          = input[offset];                                                                  \
}

#if __CUDACC_VER_MAJOR__ >= 9
    cvtSMCHANNELN4CXTONC(half)
#endif
    cvtSMCHANNELN4CXTONC(float)
    cvtSMCHANNELN4CXTONC(char)
    cvtSMCHANNELN4CXTONC(double)



#define cvtSMCHANNELNCTON4CX(type)                                                                               \
template <>                                                                                                      \
__global__ void cuda_kernel_small_channel_cvtformat<type, NDARRAY_N4CX>(                                         \
    type * input,                                                                                                \
    int num_elems,                                                                                               \
    DivModFast inner_fast,                                                                                       \
    DivModFast src_pad_fast,                                                                                     \
    DivModFast dst_pad_fast,                                                                                     \
    type* output,                                                                                                \
    ReFormatParam param)                                                                                         \
{                                                                                                                \
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;                                                       \
    if (tid >= num_elems)                                                                                        \
        return;                                                                                                  \
    int inner_idx, num_inner, c_idx;                                                                             \
    inner_fast.divmod(tid, num_inner, inner_idx);                                                                \
    src_pad_fast.divmod(num_inner, num_inner, c_idx);                                                            \
    const int c4_idx           = c_idx / 4;                                                                      \
    const int c_in_c4_idx      = c_idx % 4;                                                                      \
    const uint64_t size        = param.n_inner;                                                                  \
    const uint64_t padChannels = param.dst_pad;                                                                  \
    const uint64_t numChannels = param.channel;                                                                  \
    const uint64_t offset      = num_inner * padChannels * size + (c4_idx * size + inner_idx) * 4 + c_in_c4_idx; \
    const uint64_t inOffset    = num_inner * numChannels * size + c_idx * size + inner_idx;                      \
    output[offset]             = input[inOffset];                                                                \
}

#if __CUDACC_VER_MAJOR__ >= 9
    cvtSMCHANNELNCTON4CX(half)
#endif
    cvtSMCHANNELNCTON4CX(float)
    cvtSMCHANNELNCTON4CX(char)
    cvtSMCHANNELNCTON4CX(double)



#define MAX_DIM 65533
template<CVTFormatMode mode>
void GenDimParam(
    ReFormatParam param,
    dim3& dimBlock,
    dim3& dimGrid)
{
    dimGrid.z = param.n_outer >= MAX_DIM ? MAX_DIM : param.n_outer;
    if (mode == NHWC8_NDARRAY) {
        dimBlock.x = DIM;
        dimBlock.y = DIM;
        dimGrid.x  = DivUp(param.src_pad, DIM);
        dimGrid.y  = DivUp(param.n_inner, DIM);
    } else if (mode == NDARRAY_NHWC8) {
        dimBlock.x = DIM;
        dimBlock.y = DIM;
        dimGrid.x  = DivUp(param.n_inner, DIM);
        dimGrid.y  = DivUp(param.dst_pad, DIM);
    } else if (mode == N4CX_NDARRAY) {
        dimBlock.x = DIM;
        dimBlock.y = 1;
        dimGrid.x  = DivUp(param.n_inner, DIM);
        dimGrid.y  = param.src_pad / 4;
    } else if (mode == NDARRAY_N4CX) {
        dimBlock.x = DIM;
        dimBlock.y = 1;
        dimGrid.x  = DivUp(param.n_inner, DIM);
        dimGrid.y  = param.dst_pad / 4;
    } else {

    }
}

#define RFNHWC8                 \
    case NDARRAY_NHWC8:         \
        RUN(NDARRAY_NHWC8);     \
    case NHWC8_NDARRAY:         \
        RUN(NHWC8_NDARRAY);

#define RFN4CX             \
    case NDARRAY_N4CX:     \
        RUN(NDARRAY_N4CX); \
    case N4CX_NDARRAY:     \
        RUN(N4CX_NDARRAY);

void PPLCUDANormalCVTFormat(hipStream_t stream, const void *input, void *output, ReFormatParam param)
{
#define RUN(mode)                                                                     \
    do {                                                                              \
        dim3 dimBlock(32, 1, 1);                                                      \
        dim3 dimGrid(32, 1, 1);                                                       \
        GenDimParam<mode>(param, dimBlock, dimGrid);                                  \
        switch (GetSizeOfDataType(param.out_type)) {                                    \
            case 1:                                                                   \
                cuda_kernel_cvtformat<char, mode><<<dimGrid, dimBlock, 0, stream>>>(  \
                    (char *)input, (char *)output, param);                            \
                break;                                                                \
            case 2:                                                                   \
                cuda_kernel_cvtformat<half, mode><<<dimGrid, dimBlock, 0, stream>>>(  \
                    (half *)input, (half *)output, param);                            \
                break;                                                                \
            case 4:                                                                   \
                cuda_kernel_cvtformat<float, mode><<<dimGrid, dimBlock, 0, stream>>>( \
                    (float *)input, (float *)output, param);                          \
                break;                                                                \
            case 8:                                                                   \
                cuda_kernel_cvtformat<double, mode><<<dimGrid, dimBlock, 0, stream>>>(\
                    (double *)input, (double *)output, param);                        \
                break;                                                                \
            default:                                                                  \
                break;                                                                \
        }                                                                             \
        return;                                                                       \
    } while (0)

    switch (GetCVTFormatMode(param)) {
        RFNHWC8
        RFN4CX
        default:
            return;
    }
#undef RUN
}

void PPLCUDASmallChannelCVTFormat(hipStream_t stream, const void *input, void *output, ReFormatParam param)
{
#define RUN(mode)                                                                     \
    do {                                                                              \
        dim3 dimBlock(256, 1, 1);                                                     \
        int num_elems = param.out_elems;                                              \
        dim3 dimGrid(DivUp(num_elems, 256), 1, 1);                                    \
        DivModFast inner_fast(param.n_inner);                                         \
        DivModFast src_pad_fast(param.src_pad);                                       \
        DivModFast dst_pad_fast(param.dst_pad);                                       \
        switch (GetSizeOfDataType(param.out_type)) {                                    \
            case 1:                                                                   \
                cuda_kernel_small_channel_cvtformat<char, mode><<<dimGrid, dimBlock, 0, stream>>>(  \
                    (char *)input, num_elems, inner_fast, src_pad_fast, dst_pad_fast, \
                                    (char *)output, param);                           \
                break;                                                                \
            case 2:                                                                   \
                cuda_kernel_small_channel_cvtformat<half, mode><<<dimGrid, dimBlock, 0, stream>>>(  \
                    (half *)input, num_elems, inner_fast, src_pad_fast, dst_pad_fast, \
                                (half *)output, param);                               \
                break;                                                                \
            case 4:                                                                   \
                cuda_kernel_small_channel_cvtformat<float, mode><<<dimGrid, dimBlock, 0, stream>>>(  \
                    (float *)input, num_elems, inner_fast, src_pad_fast, dst_pad_fast, \
                                (float *)output, param);                               \
                break;                                                                \
            case 8:                                                                   \
                cuda_kernel_small_channel_cvtformat<double, mode><<<dimGrid, dimBlock, 0, stream>>>(  \
                    (double *)input, num_elems, inner_fast, src_pad_fast, dst_pad_fast, \
                                (double *)output, param);                               \
                break;                                                                \
            default:                                                                  \
                break;                                                                \
        }                                                                             \
        return;                                                                       \
    } while (0)

    switch (GetCVTFormatMode(param)) {
        RFNHWC8
        RFN4CX
        default:
            return;
    }
#undef RUN
}

void PPLCUDACVTFormat(
    hipStream_t stream,
    const void* input,
    void* output,
    ReFormatParam param)
{
    if (param.channel < LEASTCHANNEL) {
        if (param.out_type == DATATYPE_INT8) {
            auto host_in = new int8_t[param.in_elems];
            auto host_out = new int8_t[param.out_elems];
            hipMemcpy(host_in, input, param.in_elems, hipMemcpyDefault);
            PPLCUDASmallChannelCVTFormat(stream, input, output, param);
            hipMemcpy(host_out, output, param.out_elems, hipMemcpyDefault);
            delete[] host_in;
            delete[] host_out;
        } else if (param.out_type == DATATYPE_FLOAT32) {
            auto host_in = new float[param.in_elems];
            auto host_out = new float[param.out_elems];
            hipMemcpy(host_in, input, 4 * param.in_elems, hipMemcpyDefault);
            PPLCUDASmallChannelCVTFormat(stream, input, output, param);
            hipMemcpy(host_out, output, 4 * param.out_elems, hipMemcpyDefault);
            delete[] host_in;
            delete[] host_out;
        } else {
            PPLCUDASmallChannelCVTFormat(stream, input, output, param);
        }
    } else
    {
        PPLCUDANormalCVTFormat(stream, input, output, param);
    }
}
CVTFormatMode GetCVTFormatMode(ReFormatParam param)
{
    if (param.in_format == DATAFORMAT_NDARRAY) {
        switch (param.out_format) {
            case DATAFORMAT_NHWC8:
                return NDARRAY_NHWC8;
            case DATAFORMAT_N4CX:
                return NDARRAY_N4CX;
            default:
                return CVTFormatUnknown;
        }
    } else if (param.in_format == DATAFORMAT_N4CX) {
        switch (param.out_format) {
            case DATAFORMAT_NDARRAY:
                return N4CX_NDARRAY;
            default:
                return CVTFormatUnknown;
        }
    } else if (param.in_format == DATAFORMAT_NHWC8) {
        switch (param.out_format) {
            case DATAFORMAT_NDARRAY:
                return NHWC8_NDARRAY;
            default:
                return CVTFormatUnknown;
        }
    } else {
        return CVTFormatUnknown;
    }
}

CVTTypeMode GetCVTTypeMode(ReFormatParam param)
{
    if (param.in_type == DATATYPE_FLOAT32) {
        switch (param.out_type) {
            case DATATYPE_FLOAT16:
                return FLOAT32_FLOAT16;
            case DATATYPE_INT8:
                return FLOAT32_INT8;
            case DATATYPE_INT4B:
                return FLOAT32_INT4B;
            default:
                return CVTTypeUnknown;
        }
    }
    if (param.in_type == DATATYPE_FLOAT16) {
        switch (param.out_type) {
            case DATATYPE_FLOAT32:
                return FLOAT16_FLOAT32;
            case DATATYPE_INT8:
                return FLOAT16_INT8;
            case DATATYPE_INT4B:
                return FLOAT16_INT4B;
            default:
                return CVTTypeUnknown;
        }
    }
    if (param.in_type == DATATYPE_INT8) {
        switch (param.out_type) {
            case DATATYPE_FLOAT16:
                return INT8_FLOAT16;
            case DATATYPE_FLOAT32:
                return INT8_FLOAT32;
            case DATATYPE_INT4B:
                return INT8_INT4B;
            case DATATYPE_INT8:
                return INT8_INT8;
            default:
                return CVTTypeUnknown;
        }
    }
    if (param.in_type == DATATYPE_INT4B) {
        switch (param.out_type) {
            case DATATYPE_FLOAT16:
                return INT4B_FLOAT16;
            case DATATYPE_FLOAT32:
                return INT4B_FLOAT32;
            case DATATYPE_INT8:
                return INT4B_INT8;
            case DATATYPE_INT4B:
                return INT4B_INT4B;
            default:
                return CVTTypeUnknown;
        }
    }
    if (param.in_type == DATATYPE_INT32) {
        switch (param.out_type) {
            case DATATYPE_INT64:
                return INT32_INT64;
            default:
                return CVTTypeUnknown;
        }
    }
    if (param.in_type == DATATYPE_INT64) {
        switch (param.out_type) {
            case DATATYPE_INT32:
                return INT64_INT32;
            default:
                return CVTTypeUnknown;
        }
    }
    return CVTTypeUnknown;
}

bool IsFloatEqual(const std::vector<float>& a, const std::vector<float>& b) {
    if (a.size() != b.size()) {
        return false;
    }
    for (uint32_t i = 0; i < a.size(); i++) {
        if (fabs(a[0] - b[0]) > FLT_EPSILON) {
            return false;
        }
    }
    return true;
}

bool EqualQuant(const ppl::nn::cuda::CudaTensorQuant& quant_a, const ppl::nn::cuda::CudaTensorQuant& quant_b) {
    return quant_a.bit_width == quant_b.bit_width &&
           IsFloatEqual(quant_a.scale, quant_b.scale) &&
           IsFloatEqual(quant_a.zero_point, quant_b.zero_point);
}

ppl::common::RetCode SetReLayoutParam(
    ReFormatParam *param,
    const TensorShape& input,
    const TensorShape& output)
{
    param->n_outer = input.GetDim(0);
    param->channel = input.GetDimCount() > 1 ? input.GetDim(1) : 1;
    param->n_inner = input.GetDimCount() > 2 ? input.GetElementsFromDimensionIncludingPadding(2) : 1;
    param->in_format = input.GetDataFormat();
    param->out_format = output.GetDataFormat();
    param->in_type = input.GetDataType();
    param->out_type = output.GetDataType();
    param->mix_type   = (param->in_type != param->out_type);
    param->mix_format = (param->in_format != param->out_format);

    param->src_pad = Align(param->channel, AlignDataFormat(param->in_format));
    param->dst_pad = Align(param->channel, AlignDataFormat(param->out_format));

    param->out_elems = output.GetElementsIncludingPadding();
    param->in_elems = input.GetElementsIncludingPadding();
    return RC_SUCCESS;

}

ppl::common::RetCode SetReLayoutParam(
    ReFormatParam *param,
    const TensorShape& input,
    const ppl::nn::cuda::CudaTensorQuant& input_quant,
    const TensorShape& output,
    const ppl::nn::cuda::CudaTensorQuant& output_quant)
{
    SetReLayoutParam(param, input, output);
    param->i_step = input_quant.scale[0];
    param->i_zero_point = input_quant.zero_point[0];
    param->o_step = output_quant.scale[0];
    param->o_zero_point = output_quant.zero_point[0];
    if (param->in_type == param->out_type) {
        param->mix_type = !EqualQuant(input_quant, output_quant);
    }
    return RC_SUCCESS;
}

void PPLCUDADataConvert(
    hipStream_t stream,
    const void* input,
    void* output,
    void* tempBuf,
    ReFormatParam& param)
{
    if (param.in_format != param.out_format && param.in_type != param.out_type) {
        PPLCUDACVTTypePerTensor(stream, input, tempBuf, param);
        PPLCUDACVTFormat(stream, tempBuf, output, param);
        return;
    } else if (param.in_format != param.out_format) {
        PPLCUDACVTFormat(stream, input, output, param);
        return;
    } else if (param.in_type != param.out_type) {
        PPLCUDACVTTypePerTensor(stream, input, output, param);
        return;
    } else {
        return;
    }
}
