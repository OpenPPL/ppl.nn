#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include <float.h>
#include <iostream>
#include "cudakernel/reformat/reformat.h"
#include "cudakernel/common/common.h"
#include "cudakernel/common/divmod_fast.h"
#include "cudakernel/common/macro.h"

#include "hip/hip_fp16.h"
using namespace PPLCUDA;
using namespace ppl::nn;
using namespace ppl::common;

#define DIM 32
#define LEASTCHANNEL 16
template <typename T, CVTFormatMode mode>
__global__ void cuda_kernel_cvtformat(
    T* input,
    T* output,
    ReFormatParam param)
{
}


#define cvtC16TOC8(type)                                                                               \
template<>                                                                                              \
__global__ void cuda_kernel_cvtformat<type, NHWC16_NHWC8>(                                              \
    type* input,                                                                                        \
    type* output,                                                                                       \
    ReFormatParam param)                                                                                \
{                                                                                                       \
                                                                                                        \
    int64_t num = blockIdx.z;                                                                           \
    for (int n = num; n < param.n_outer; n+= blockDim.z) {                                              \
        int64_t idx_w = blockIdx.x * blockDim.x + threadIdx.x;                                          \
        int64_t idx_h = blockIdx.y * blockDim.y + threadIdx.y;                                          \
                                                                                                        \
        if (idx_w < param.dst_pad && idx_h < param.n_inner) {                                           \
            int64_t dst_offset = n * param.dst_pad * param.n_inner + idx_h * param.dst_pad + idx_w;     \
            int64_t src_offset = n * param.src_pad * param.n_inner + idx_h * param.src_pad + idx_w;     \
            output[dst_offset] = input[src_offset];                   \
        }                                                                                               \
    }                                                                                                   \
}                                                                                                       

#if __CUDACC_VER_MAJOR__ >= 9
    cvtC16TOC8(half)
#endif
    cvtC16TOC8(float)
    cvtC16TOC8(char)
    cvtC16TOC8(double)
    cvtC16TOC8(int8_t)



#define cvtC8TOC16(type)                                                                               \
template<>                                                                                              \
__global__ void cuda_kernel_cvtformat<type, NHWC8_NHWC16>(                                              \
    type* input,                                                                                        \
    type* output,                                                                                       \
    ReFormatParam param)                                                                                \
{                                                                                                       \
                                                                                                        \
    int64_t num = blockIdx.z;                                                                           \
    for (int n = num; n < param.n_outer; n+= blockDim.z) {                                              \
        int64_t idx_w = blockIdx.x * blockDim.x + threadIdx.x;                                          \
        int64_t idx_h = blockIdx.y * blockDim.y + threadIdx.y;                                          \
                                                                                                        \
        if (idx_w < param.dst_pad && idx_h < param.n_inner) {                                           \
            int64_t dst_offset = n * param.dst_pad * param.n_inner + idx_h * param.dst_pad + idx_w;     \
            int64_t src_offset = n * param.src_pad * param.n_inner + idx_h * param.src_pad + idx_w;     \
            output[dst_offset] = idx_w < param.src_pad ? input[src_offset] : type(0);                   \
        }                                                                                               \
    }                                                                                                   \
}                                                                                                       

#if __CUDACC_VER_MAJOR__ >= 9
    cvtC8TOC16(half)
#endif
    cvtC8TOC16(float)
    cvtC8TOC16(char)
    cvtC8TOC16(double)
    cvtC8TOC16(int8_t)

#define cvtNCTONHWC(type)                                                                               \
template<>                                                                                              \
__global__ void cuda_kernel_cvtformat<type, NDARRAY_NHWC>(                                              \
    type* input,                                                                                        \
    type* output,                                                                                       \
    ReFormatParam param)                                                                                \
{                                                                                                       \
    __shared__ type share_val[DIM][DIM + 1];                                                            \
                                                                                                        \
    int64_t num = blockIdx.z;                                                                           \
    for (int n = num; n < param.n_outer; n+= blockDim.z) {                                              \
        int64_t idx_w = blockIdx.x * blockDim.x + threadIdx.x;                                          \
        int64_t idx_h = blockIdx.y * blockDim.y + threadIdx.y;                                          \
                                                                                                        \
        if (idx_w < param.n_inner && idx_h < param.src_pad) {                                           \
            int64_t offset = n * param.src_pad * param.n_inner + idx_h * param.n_inner + idx_w;         \
            share_val[threadIdx.y][threadIdx.x] = input[offset];                                        \
        } else {                                                                                        \
            share_val[threadIdx.y][threadIdx.x] = (type)0;                                              \
        }                                                                                               \
        __syncthreads();                                                                                \
                                                                                                        \
        idx_w = blockIdx.y * blockDim.y + threadIdx.x;                                                  \
        idx_h = blockIdx.x * blockDim.x + threadIdx.y;                                                  \
                                                                                                        \
        if (idx_w < param.dst_pad && idx_h < param.n_inner) {                                           \
            int64_t offset = n * param.dst_pad * param.n_inner + idx_h * param.dst_pad + idx_w;         \
            output[offset] = share_val[threadIdx.x][threadIdx.y];                                       \
        }                                                                                               \
    }                                                                                                   \
}

#if __CUDACC_VER_MAJOR__ >= 9
    cvtNCTONHWC(half)
#endif
    cvtNCTONHWC(float)
    cvtNCTONHWC(char)
    cvtNCTONHWC(double)
    cvtNCTONHWC(int8_t)



#define cvtNHWC8TONC(type)                                                                               \
template<>                                                                                              \
__global__ void cuda_kernel_cvtformat<type, NHWC_NDARRAY>(                                              \
    type* input,                                                                                        \
    type* output,                                                                                       \
    ReFormatParam param)                                                                                \
{                                                                                                       \
    __shared__ type share_val[DIM][DIM + 1];                                                            \
                                                                                                        \
    int64_t num = blockIdx.z;                                                                           \
    for (int n = num; n < param.n_outer; n += blockDim.z) {                                              \
        int64_t idx_w = blockIdx.x * blockDim.x + threadIdx.x;                                          \
        int64_t idx_h = blockIdx.y * blockDim.y + threadIdx.y;                                          \
                                                                                                        \
        if (idx_w < param.src_pad && idx_h < param.n_inner) {                                           \
            int64_t offset = n * param.src_pad * param.n_inner + idx_h * param.src_pad + idx_w;         \
            share_val[threadIdx.y][threadIdx.x] = input[offset];                                        \
        } else {                                                                                        \
            share_val[threadIdx.y][threadIdx.x] = (type)0;                                              \
        }                                                                                               \
        __syncthreads();                                                                                \
                                                                                                        \
        idx_w = blockIdx.y * blockDim.y + threadIdx.x;                                                  \
        idx_h = blockIdx.x * blockDim.x + threadIdx.y;                                                  \
                                                                                                        \
        if (idx_w < param.n_inner && idx_h < param.dst_pad) {                                           \
            int64_t offset = n * param.dst_pad * param.n_inner + idx_h * param.n_inner + idx_w;         \
            output[offset] = share_val[threadIdx.x][threadIdx.y];                                       \
        }                                                                                               \
    }                                                                                                   \
}

#if __CUDACC_VER_MAJOR__ >= 9
    cvtNHWC8TONC(half)
#endif
    cvtNHWC8TONC(float)
    cvtNHWC8TONC(char)
    cvtNHWC8TONC(double)
    cvtNHWC8TONC(int8_t)



#define cvtN4CXTONC(type)                                                                                              \
template <>                                                                                                            \
__global__ void cuda_kernel_cvtformat<type, N4CX_NDARRAY>(                                                             \
    type * input,                                                                                                      \
    type * output,                                                                                                     \
    ReFormatParam param)                                                                                               \
{                                                                                                                      \
    const uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;                                                        \
    if (tid >= param.n_inner)                                                                                          \
        return;                                                                                                        \
    const uint64_t inner_idx = tid;                                                                                    \
    const uint64_t num_inner = blockIdx.z;                                                                             \
    const uint64_t c4_idx    = blockIdx.y;                                                                             \
    _Pragma("unroll 4") for (int c_in_c4_idx = 0; c_in_c4_idx < 4; c_in_c4_idx++)                                      \
    {                                                                                                                  \
        const uint64_t c_idx       = c4_idx * 4 + c_in_c4_idx;                                                         \
        const uint64_t size        = param.n_inner;                                                                    \
        const uint64_t padChannels = gridDim.y * 4;                                                                    \
        const uint64_t numChannels = param.channel;                                                                    \
        if (c_idx < numChannels) {                                                                                     \
            const uint64_t offset    = num_inner * padChannels * size + (c4_idx * size + inner_idx) * 4 + c_in_c4_idx; \
            const uint64_t outOffset = num_inner * numChannels * size + c_idx * size + inner_idx;                      \
            output[outOffset]        = input[offset];                                                                  \
        }                                                                                                              \
    }                                                                                                                  \
}

#if __CUDACC_VER_MAJOR__ >= 9
    cvtN4CXTONC(half)
#endif
    cvtN4CXTONC(float)
    cvtN4CXTONC(char)
    cvtN4CXTONC(double)
    cvtN4CXTONC(int8_t)



#define cvtNCTON4CX(type)                                                                                             \
template <>                                                                                                           \
__global__ void cuda_kernel_cvtformat<type, NDARRAY_N4CX>(                                                            \
    type * input,                                                                                                     \
    type * output,                                                                                                    \
    ReFormatParam param)                                                                                              \
{                                                                                                                     \
    const uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;                                                       \
    if (tid >= param.n_inner)                                                                                         \
        return;                                                                                                       \
    const uint64_t inner_idx = tid;                                                                                   \
    const uint64_t num_inner = blockIdx.z;                                                                            \
    const uint64_t c4_idx    = blockIdx.y;                                                                            \
    _Pragma("unroll 4") for (int c_in_c4_idx = 0; c_in_c4_idx < 4; c_in_c4_idx++)                                     \
    {                                                                                                                 \
        const uint64_t c_idx       = c4_idx * 4 + c_in_c4_idx;                                                        \
        const uint64_t size        = param.n_inner;                                                                   \
        const uint64_t padChannels = gridDim.y * 4;                                                                   \
        const uint64_t numChannels = param.channel;                                                                   \
        if (c_idx < numChannels) {                                                                                    \
            const uint64_t offset   = num_inner * padChannels * size + (c4_idx * size + inner_idx) * 4 + c_in_c4_idx; \
            const uint64_t inOffset = num_inner * numChannels * size + c_idx * size + inner_idx;                      \
            output[offset]          = input[inOffset];                                                                \
        }                                                                                                             \
    }                                                                                                                 \
}

#if __CUDACC_VER_MAJOR__ >= 9
    cvtNCTON4CX(half)
#endif
    cvtNCTON4CX(float)
    cvtNCTON4CX(char)
    cvtNCTON4CX(double)
    cvtNCTON4CX(int8_t)



template <typename T, CVTFormatMode mode>
__global__ void cuda_kernel_small_channel_cvtformat(
    T* input,
    int num_elems,
    DivModFast inner_fast,
    DivModFast src_pad_fast,
    DivModFast dst_pad_fast,
    T* output,
    ReFormatParam param)
{
}

#define cvtSMCHANNELNCTONHWC8(type)                                                                      \
template<>                                                                                              \
__global__ void cuda_kernel_small_channel_cvtformat<type, NDARRAY_NHWC>(                                \
    type* input,                                                                                        \
    int num_elems,                                                                                      \
    DivModFast inner_fast,                                                                              \
    DivModFast src_pad_fast,                                                                            \
    DivModFast dst_pad_fast,                                                                            \
    type* output,                                                                                       \
    ReFormatParam param)                                                                                \
{                                                                                                       \
    int tid = blockIdx.x * blockDim.x + threadIdx.x;                                                    \
    if (tid >= num_elems) return;                                                                       \
    int inner_idx = 0, num_inner = 0, c_idx = 0;                                                        \
    dst_pad_fast.divmod(tid, num_inner, c_idx);                                                         \
    inner_idx = inner_fast.mod(num_inner);                                                              \
    int outer_idx = inner_fast.div(num_inner);                                                              \
    int offset = outer_idx * param.src_pad * param.n_inner + c_idx * param.n_inner + inner_idx;         \
    output[tid] =  c_idx < param.src_pad ? input[offset] : (type)0;                                     \
}

#if __CUDACC_VER_MAJOR__ >= 9
    cvtSMCHANNELNCTONHWC8(half)
#endif
    cvtSMCHANNELNCTONHWC8(float)
    cvtSMCHANNELNCTONHWC8(char)
    cvtSMCHANNELNCTONHWC8(double)
    cvtSMCHANNELNCTONHWC8(int8_t)



#define cvtSMCHANNELNHWC8TONC(type)                                                                      \
template<>                                                                                              \
__global__ void cuda_kernel_small_channel_cvtformat<type, NHWC_NDARRAY>(                                \
    type* input,                                                                                        \
    int num_elems,                                                                                      \
    DivModFast inner_fast,                                                                              \
    DivModFast src_pad_fast,                                                                            \
    DivModFast dst_pad_fast,                                                                            \
    type* output,                                                                                       \
    ReFormatParam param)                                                                                \
{                                                                                                       \
    int tid = blockIdx.x * blockDim.x + threadIdx.x;                                                    \
    if (tid >= num_elems) return;                                                                       \
    int inner_idx = 0, num_inner = 0, c_idx = 0;                                                        \
    inner_fast.divmod(tid, num_inner, inner_idx);                                                       \
    c_idx = dst_pad_fast.mod(num_inner);                                                                \
    int outer_idx = tid / (param.dst_pad * param.n_inner);                                              \
    int offset = outer_idx * param.src_pad * param.n_inner + c_idx + inner_idx * param.src_pad;         \
    output[tid] = input[offset];                                                                        \
}

#if __CUDACC_VER_MAJOR__ >= 9
    cvtSMCHANNELNHWC8TONC(half)
#endif
    cvtSMCHANNELNHWC8TONC(float)
    cvtSMCHANNELNHWC8TONC(char)
    cvtSMCHANNELNHWC8TONC(double)
    cvtSMCHANNELNHWC8TONC(int8_t)



#define cvtSMCHANNELN4CXTONC(type)                                                                               \
template <>                                                                                                      \
__global__ void cuda_kernel_small_channel_cvtformat<type, N4CX_NDARRAY>(                                         \
    type * input,                                                                                                \
    int num_elems,                                                                                               \
    DivModFast inner_fast,                                                                                       \
    DivModFast src_pad_fast,                                                                                     \
    DivModFast dst_pad_fast,                                                                                     \
    type* output,                                                                                                \
    ReFormatParam param)                                                                                         \
{                                                                                                                \
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;                                                       \
    if (tid >= num_elems)                                                                                        \
        return;                                                                                                  \
    int inner_idx, num_inner, c_idx;                                                                             \
    inner_fast.divmod(tid, num_inner, inner_idx);                                                                \
    src_pad_fast.divmod(num_inner, num_inner, c_idx);                                                            \
    const int c4_idx           = c_idx / 4;                                                                      \
    const int c_in_c4_idx      = c_idx % 4;                                                                      \
    const uint64_t size        = param.n_inner;                                                                  \
    const uint64_t padChannels = param.src_pad;                                                                  \
    const uint64_t numChannels = param.channel;                                                                  \
    const uint64_t offset      = num_inner * padChannels * size + (c4_idx * size + inner_idx) * 4 + c_in_c4_idx; \
    const uint64_t outOffset   = num_inner * numChannels * size + c_idx * size + inner_idx;                      \
    output[outOffset]          = input[offset];                                                                  \
}

#if __CUDACC_VER_MAJOR__ >= 9
    cvtSMCHANNELN4CXTONC(half)
#endif
    cvtSMCHANNELN4CXTONC(float)
    cvtSMCHANNELN4CXTONC(char)
    cvtSMCHANNELN4CXTONC(double)
    cvtSMCHANNELN4CXTONC(int8_t)



#define cvtSMCHANNELNCTON4CX(type)                                                                               \
template <>                                                                                                      \
__global__ void cuda_kernel_small_channel_cvtformat<type, NDARRAY_N4CX>(                                         \
    type * input,                                                                                                \
    int num_elems,                                                                                               \
    DivModFast inner_fast,                                                                                       \
    DivModFast src_pad_fast,                                                                                     \
    DivModFast dst_pad_fast,                                                                                     \
    type* output,                                                                                                \
    ReFormatParam param)                                                                                         \
{                                                                                                                \
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;                                                       \
    if (tid >= num_elems)                                                                                        \
        return;                                                                                                  \
    int inner_idx, num_inner, c_idx;                                                                             \
    inner_fast.divmod(tid, num_inner, inner_idx);                                                                \
    src_pad_fast.divmod(num_inner, num_inner, c_idx);                                                            \
    const int c4_idx           = c_idx / 4;                                                                      \
    const int c_in_c4_idx      = c_idx % 4;                                                                      \
    const uint64_t size        = param.n_inner;                                                                  \
    const uint64_t padChannels = param.dst_pad;                                                                  \
    const uint64_t numChannels = param.channel;                                                                  \
    const uint64_t offset      = num_inner * padChannels * size + (c4_idx * size + inner_idx) * 4 + c_in_c4_idx; \
    const uint64_t inOffset    = num_inner * numChannels * size + c_idx * size + inner_idx;                      \
    output[offset]             = input[inOffset];                                                                \
}

#if __CUDACC_VER_MAJOR__ >= 9
    cvtSMCHANNELNCTON4CX(half)
#endif
    cvtSMCHANNELNCTON4CX(float)
    cvtSMCHANNELNCTON4CX(char)
    cvtSMCHANNELNCTON4CX(double)
    cvtSMCHANNELNCTON4CX(int8_t)



#define MAX_DIM 65533
template<CVTFormatMode mode>
void GenDimParam(
    ReFormatParam param,
    dim3& dimBlock,
    dim3& dimGrid)
{
    dimGrid.z = param.n_outer >= MAX_DIM ? MAX_DIM : param.n_outer;
    if (mode == NHWC_NDARRAY) {
        dimBlock.x = DIM;
        dimBlock.y = DIM;
        dimGrid.x  = DivUp(param.src_pad, DIM);
        dimGrid.y  = DivUp(param.n_inner, DIM);
    } else if (mode == NDARRAY_NHWC) {
        dimBlock.x = DIM;
        dimBlock.y = DIM;
        dimGrid.x  = DivUp(param.n_inner, DIM);
        dimGrid.y  = DivUp(param.dst_pad, DIM);
    } else if (mode == N4CX_NDARRAY) {
        dimBlock.x = DIM;
        dimBlock.y = 1;
        dimGrid.x  = DivUp(param.n_inner, DIM);
        dimGrid.y  = param.src_pad / 4;
    } else if (mode == NDARRAY_N4CX) {
        dimBlock.x = DIM;
        dimBlock.y = 1;
        dimGrid.x  = DivUp(param.n_inner, DIM);
        dimGrid.y  = param.dst_pad / 4;
    } else if (mode == NHWC8_NHWC16){
        dimBlock.x = DIM;
        dimBlock.y = DIM;
        dimGrid.x  = DivUp(param.dst_pad, DIM);
        dimGrid.y  = DivUp(param.n_inner, DIM);
    } else if (mode == NHWC16_NHWC8){
        dimBlock.x = DIM;
        dimBlock.y = DIM;
        dimGrid.x  = DivUp(param.dst_pad, DIM);
        dimGrid.y  = DivUp(param.n_inner, DIM);
    } 
}
#define RFC8C16              \
    case NHWC8_NHWC16:         \
        RUN(NHWC8_NHWC16);     \
    case NHWC16_NHWC8:         \
        RUN(NHWC16_NHWC8);

#define RFNHWC                 \
    case NDARRAY_NHWC:         \
        RUN(NDARRAY_NHWC);     \
    case NHWC_NDARRAY:         \
        RUN(NHWC_NDARRAY);

#define RFN4CX             \
    case NDARRAY_N4CX:     \
        RUN(NDARRAY_N4CX); \
    case N4CX_NDARRAY:     \
        RUN(N4CX_NDARRAY);

void PPLCUDANormalCVTFormat(hipStream_t stream, const void *input, void *output, ReFormatParam param)
{
#define RUN(mode)                                                                     \
    do {                                                                              \
        dim3 dimBlock(32, 1, 1);                                                      \
        dim3 dimGrid(32, 1, 1);                                                       \
        GenDimParam<mode>(param, dimBlock, dimGrid);                                  \
        switch (GetSizeOfDataType(param.out_type)) {                                    \
            case 1:                                                                   \
                cuda_kernel_cvtformat<int8_t, mode><<<dimGrid, dimBlock, 0, stream>>>(  \
                    (int8_t *)input, (int8_t *)output, param);                            \
                break;                                                                \
            case 2:                                                                   \
                cuda_kernel_cvtformat<half, mode><<<dimGrid, dimBlock, 0, stream>>>(  \
                    (half *)input, (half *)output, param);                            \
                break;                                                                \
            case 4:                                                                   \
                cuda_kernel_cvtformat<float, mode><<<dimGrid, dimBlock, 0, stream>>>( \
                    (float *)input, (float *)output, param);                          \
                break;                                                                \
            case 8:                                                                   \
                cuda_kernel_cvtformat<double, mode><<<dimGrid, dimBlock, 0, stream>>>(\
                    (double *)input, (double *)output, param);                        \
                break;                                                                \
            default:                                                                  \
                break;                                                                \
        }                                                                             \
        return;                                                                       \
    } while (0)

    switch (GetCVTFormatMode(param)) {
        RFC8C16
        RFNHWC
        RFN4CX
        default:
            return;
    }
#undef RUN
}

__global__ void cuda_kernel_packed_cvtformat(
    int8_t *input,
    int8_t *output,
    DivModFast inner_fast,
    int num_elems,
    ReFormatParam param) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_elems) return;
    char val[16];
    _Pragma("unroll")
    for (int i = 0; i < 16; i++) {
        val[i] = 0;
    }
    int b = 0, hw_idx = 0;
    inner_fast.divmod(tid, b, hw_idx);
    int offset = b * param.n_inner * param.src_pad + hw_idx;
    for (int i = 0; i < param.src_pad; i++) {
        val[i] = input[offset];
        offset += param.n_inner;
    }
    float4* dst = (float4*)val;
    float4* dst_out = (float4*)output;
    dst_out[tid] = dst[0];
}
void PPLCUDASmallChannelCVTPackedFormat(hipStream_t stream, const void *input, void *output, ReFormatParam param)
{
    dim3 dimBlock(256, 1, 1);
    int num_elems = param.out_elems / param.dst_pad;
    dim3 dimGrid(DivUp(num_elems, 256), 1, 1);
    DivModFast inner_fast(param.n_inner);
    cuda_kernel_packed_cvtformat<<<dimGrid, dimBlock, 0, stream>>>((int8_t*)input, (int8_t*)output, inner_fast, num_elems, param);
    
}
void PPLCUDASmallChannelCVTFormat(hipStream_t stream, const void *input, void *output, ReFormatParam param)
{
    if (param.out_type == ppl::common::DATATYPE_INT8 && param.out_format == ppl::common::DATAFORMAT_NHWC16
        && param.in_format == ppl::common::DATAFORMAT_NDARRAY) {
            PPLCUDASmallChannelCVTPackedFormat(stream, input, output, param);
            return; 
        }
#define RUN(mode)                                                                     \
    do {                                                                              \
        dim3 dimBlock(256, 1, 1);                                                     \
        int num_elems = param.out_elems;                                              \
        dim3 dimGrid(DivUp(num_elems, 256), 1, 1);                                    \
        DivModFast inner_fast(param.n_inner);                                         \
        DivModFast src_pad_fast(param.src_pad);                                       \
        DivModFast dst_pad_fast(param.dst_pad);                                       \
        switch (GetSizeOfDataType(param.out_type)) {                                    \
            case 1:                                                                   \
                cuda_kernel_small_channel_cvtformat<char, mode><<<dimGrid, dimBlock, 0, stream>>>(  \
                    (char *)input, num_elems, inner_fast, src_pad_fast, dst_pad_fast, \
                                    (char *)output, param);                           \
                break;                                                                \
            case 2:                                                                   \
                cuda_kernel_small_channel_cvtformat<half, mode><<<dimGrid, dimBlock, 0, stream>>>(  \
                    (half *)input, num_elems, inner_fast, src_pad_fast, dst_pad_fast, \
                                (half *)output, param);                               \
                break;                                                                \
            case 4:                                                                   \
                cuda_kernel_small_channel_cvtformat<float, mode><<<dimGrid, dimBlock, 0, stream>>>(  \
                    (float *)input, num_elems, inner_fast, src_pad_fast, dst_pad_fast, \
                                (float *)output, param);                               \
                break;                                                                \
            case 8:                                                                   \
                cuda_kernel_small_channel_cvtformat<double, mode><<<dimGrid, dimBlock, 0, stream>>>(  \
                    (double *)input, num_elems, inner_fast, src_pad_fast, dst_pad_fast, \
                                (double *)output, param);                               \
                break;                                                                \
            default:                                                                  \
                break;                                                                \
        }                                                                             \
        return;                                                                       \
    } while (0)

    switch (GetCVTFormatMode(param)) {
        RFNHWC
        RFN4CX
        default:
            return;
    }
#undef RUN
}

void PPLCUDACVTFormat(
    hipStream_t stream,
    const void* input,
    void* output,
    ReFormatParam param)
{
    if (param.channel < LEASTCHANNEL) {
        if (param.out_type == DATATYPE_INT8) {
            PPLCUDASmallChannelCVTFormat(stream, input, output, param);
        } else if (param.out_type == DATATYPE_FLOAT32) {
            PPLCUDASmallChannelCVTFormat(stream, input, output, param);
        } else {
            PPLCUDASmallChannelCVTFormat(stream, input, output, param);
        }
    } else
    {
        PPLCUDANormalCVTFormat(stream, input, output, param);
    }
}
CVTFormatMode GetCVTFormatMode(ReFormatParam param)
{
    if (param.in_format == DATAFORMAT_NDARRAY) {
        switch (param.out_format) {
            case DATAFORMAT_NHWC8:
                return NDARRAY_NHWC;
            case DATAFORMAT_NHWC16:
                return NDARRAY_NHWC;
            case DATAFORMAT_N4CX:
                return NDARRAY_N4CX;
            default:
                return CVTFormatUnknown;
        }
    } else if (param.in_format == DATAFORMAT_N4CX) {
        switch (param.out_format) {
            case DATAFORMAT_NDARRAY:
                return N4CX_NDARRAY;
            default:
                return CVTFormatUnknown;
        }
    } else if (param.in_format == DATAFORMAT_NHWC8) {
        switch (param.out_format) {
            case DATAFORMAT_NDARRAY:
                return NHWC_NDARRAY;
            case DATAFORMAT_NHWC16:
                return NHWC8_NHWC16;
            default:
                return CVTFormatUnknown;
        }
    } else if (param.in_format == DATAFORMAT_NHWC16) {
        switch (param.out_format) {
            case DATAFORMAT_NDARRAY:
                return NHWC_NDARRAY;
            case DATAFORMAT_NHWC8:
                return NHWC16_NHWC8;
            default:
                return CVTFormatUnknown;
        }
    } else {
        return CVTFormatUnknown;
    }
}

CVTTypeMode GetCVTTypeMode(ReFormatParam param)
{
    if (param.in_type == DATATYPE_FLOAT32) {
        switch (param.out_type) {
            case DATATYPE_FLOAT16:
                return FLOAT32_FLOAT16;
            case DATATYPE_INT8:
                return FLOAT32_INT8;
            case DATATYPE_INT4B:
                return FLOAT32_INT4B;
            default:
                return CVTTypeUnknown;
        }
    }
    if (param.in_type == DATATYPE_FLOAT16) {
        switch (param.out_type) {
            case DATATYPE_FLOAT32:
                return FLOAT16_FLOAT32;
            case DATATYPE_INT8:
                return FLOAT16_INT8;
            case DATATYPE_INT4B:
                return FLOAT16_INT4B;
            default:
                return CVTTypeUnknown;
        }
    }
    if (param.in_type == DATATYPE_INT8) {
        switch (param.out_type) {
            case DATATYPE_FLOAT16:
                return INT8_FLOAT16;
            case DATATYPE_FLOAT32:
                return INT8_FLOAT32;
            case DATATYPE_INT4B:
                return INT8_INT4B;
            case DATATYPE_INT8:
                return INT8_INT8;
            default:
                return CVTTypeUnknown;
        }
    }
    if (param.in_type == DATATYPE_INT4B) {
        switch (param.out_type) {
            case DATATYPE_FLOAT16:
                return INT4B_FLOAT16;
            case DATATYPE_FLOAT32:
                return INT4B_FLOAT32;
            case DATATYPE_INT8:
                return INT4B_INT8;
            case DATATYPE_INT4B:
                return INT4B_INT4B;
            default:
                return CVTTypeUnknown;
        }
    }
    if (param.in_type == DATATYPE_INT32) {
        switch (param.out_type) {
            case DATATYPE_INT64:
                return INT32_INT64;
            default:
                return CVTTypeUnknown;
        }
    }
    if (param.in_type == DATATYPE_INT64) {
        switch (param.out_type) {
            case DATATYPE_INT32:
                return INT64_INT32;
            default:
                return CVTTypeUnknown;
        }
    }
    return CVTTypeUnknown;
}

bool IsFloatEqual(const std::vector<float>& a, const std::vector<float>& b) {
    if (a.size() != b.size()) {
        return false;
    }
    for (uint32_t i = 0; i < a.size(); i++) {
        if (fabs(a[i] - b[i]) > FLT_EPSILON) {
            return false;
        }
    }
    return true;
}

bool EqualQuant(const ppl::nn::cuda::CudaTensorQuant& quant_a, const ppl::nn::cuda::CudaTensorQuant& quant_b) {
    return quant_a.bit_width == quant_b.bit_width &&
           IsFloatEqual(quant_a.scale, quant_b.scale) &&
           IsFloatEqual(quant_a.zero_point, quant_b.zero_point);
}

ppl::common::RetCode SetReLayoutParam(
    ReFormatParam *param,
    const TensorShape& input,
    const TensorShape& output)
{
    if (input.GetDimCount() <= 1 &&
        ((input.GetDataFormat() == DATAFORMAT_NHWC8) ||
        (output.GetDataFormat() == DATAFORMAT_NHWC8) || 
        (input.GetDataFormat() == DATAFORMAT_NHWC16) ||
        (output.GetDataFormat() == DATAFORMAT_NHWC16)))
        return RC_INVALID_VALUE;
    param->n_outer = input.GetDim(0);
    param->channel = input.GetDimCount() > 1 ? input.GetDim(1) : 1;
    param->n_inner = input.GetDimCount() > 2 ? input.GetElementsFromDimensionIncludingPadding(2) : 1;
    param->in_format = input.GetDataFormat();
    param->out_format = output.GetDataFormat();
    param->in_type = input.GetDataType();
    param->out_type = output.GetDataType();
    param->mix_type   = (param->in_type != param->out_type);
    param->mix_format = (param->in_format != param->out_format);

    param->src_pad = Align(param->channel, AlignDataFormat(param->in_format));
    param->dst_pad = Align(param->channel, AlignDataFormat(param->out_format));

    param->out_elems = output.GetElementsIncludingPadding();
    param->in_elems = input.GetElementsIncludingPadding();
    return RC_SUCCESS;

}

ppl::common::RetCode SetReLayoutParam(
    ReFormatParam *param,
    const TensorShape& input,
    const ppl::nn::cuda::CudaTensorQuant& input_quant,
    const TensorShape& output,
    const ppl::nn::cuda::CudaTensorQuant& output_quant)
{
    SetReLayoutParam(param, input, output);
    param->same_scale = IsFloatEqual(input_quant.scale, output_quant.scale);
    if (input_quant.per_channel) {
        param->per_channel = true;
        param->quant_stride = input.GetDataFormat() == DATAFORMAT_NDARRAY? param->n_inner : 1;
        param->quant_dim_size = param->n_outer;
        param->quant_stride *= param->channel;
    } else {
        param->i_step = input_quant.scale[0];
        param->o_step = output_quant.scale[0];
    }
    param->i_zero_point = input_quant.zero_point[0];
    param->o_zero_point = output_quant.zero_point[0];
    if (param->in_type == param->out_type && param->in_type == DATATYPE_INT8) {
        param->mix_type = !EqualQuant(input_quant, output_quant);
    }
    return RC_SUCCESS;
}

void PPLCUDADataConvert(
    hipStream_t stream,
    const void* input,
    void* output,
    void* tempBuf,
    ReFormatParam& param)
{
    if (param.in_format != param.out_format && (param.in_type != param.out_type || !param.same_scale)) {
        if (param.per_channel) {
            PPLCUDACVTTypePerChannel(stream, input, tempBuf, param);
        } else {
            PPLCUDACVTTypePerTensor(stream, input, tempBuf, param);
        }
        PPLCUDACVTFormat(stream, tempBuf, output, param);
        return;
    } else if (param.in_format != param.out_format && (param.in_type = param.out_type && param.same_scale)) {
        PPLCUDACVTFormat(stream, input, output, param);
        return;
    } else if (param.in_type != param.out_type || !param.same_scale) {
        if (param.per_channel) {
            PPLCUDACVTTypePerChannel(stream, input, output, param);
        } else {
            PPLCUDACVTTypePerTensor(stream, input, output, param);
        }
        return;
    } else {
        return;
    }
}