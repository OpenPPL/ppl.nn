#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "cudakernel/memory/scatter_nd.h"
#include "cudakernel/common/divmod_fast.h"
#include "ppl/nn/common/tensor_shape.h"
#include "ppl/common/retcode.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include <vector>

template <typename T>
__global__ void ppl_cukernel_scatter_nd(
    int64_t num_elems,
    DivModFast piece_size_fast,
    int64_t* piece_offsets,
    const T* updates,
    T* output)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_elems)
        return;
    int piece_idx, offset;
    piece_size_fast.divmod(index, piece_idx, offset);
    int64_t base_offset          = piece_offsets[piece_idx];
    output[base_offset + offset] = updates[index];
}

template <typename IndexT>
__global__ void ppl_cukernel_scatter_nd_offset(
    int64_t num_pieces,
    int64_t* input_dims_gpu,
    int64_t* input_strides_gpu,
    int indices_last_dim_size,
    const IndexT* indices_data,
    int64_t* piece_offsets)
{
    int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_pieces)
        return;
    // offset
    const IndexT* indices_ptr = indices_data + index * indices_last_dim_size;
    int64_t rel_offset        = 0;
    for (int it = 0; it < indices_last_dim_size; ++it) {
        IndexT cor_val = indices_ptr[it];
        if (cor_val < 0)
            cor_val += input_dims_gpu[it];
        assert(cor_val >= 0 && cor_val < input_dims_gpu[it]);
        rel_offset += cor_val * input_strides_gpu[it];
    }
    piece_offsets[index] = rel_offset;
}

int64_t PPLScatterNDGetTempBufferSize(
    const ppl::nn::TensorShape* input_shape,
    const void* input,
    const ppl::nn::TensorShape* indices_shape,
    const void* indices)
{
    int num_input_dim   = input_shape->GetDimCount();
    int num_indices_dim = indices_shape->GetDimCount();
    int num_pieces      = indices_shape->GetElementsToDimensionIncludingPadding(num_indices_dim - 1);
    // pieces offsets and input strides and input_dims
    int64_t total_size  = (num_pieces + 2 * num_input_dim) * sizeof(int64_t);
    return total_size;
}

ppl::common::RetCode PPLCUDAScatterNDForwardImp(
    hipStream_t stream,
    const ppl::nn::TensorShape* input_shape,
    const void* input,
    const ppl::nn::TensorShape* indices_shape,
    const void* indices,
    const ppl::nn::TensorShape* updates_shape,
    const void* updates,
    const ppl::nn::TensorShape* output_shape,
    void* output,
    void* temp_buffer)
{
    int64_t num_elems_output = output_shape->GetElementsIncludingPadding();
    hipMemcpyAsync(output, input, ppl::common::GetSizeOfDataType(input_shape->GetDataType()) * num_elems_output, hipMemcpyDeviceToDevice, stream);

    int num_indices_dim       = indices_shape->GetDimCount();
    int num_input_dim         = input_shape->GetDimCount();
    int indices_last_dim_size = indices_shape->GetDim(num_indices_dim - 1);
    int num_pieces            = indices_shape->GetElementsToDimensionIncludingPadding(num_indices_dim - 1);
    int piece_size            = input_shape->GetElementsFromDimensionIncludingPadding(
        indices_last_dim_size);
    int block_size             = 256;
    // step 1: calcalute each piece's offset first
    int64_t* piece_offsets     = static_cast<int64_t*>(temp_buffer);
    int64_t* input_strides_gpu = piece_offsets + num_pieces;
    int64_t* input_dims_gpu    = input_strides_gpu + num_input_dim;
    std::vector<int64_t> input_strides(indices_last_dim_size);
    std::vector<int64_t> input_dims(num_input_dim);
    // dimension is partitioned as indices_last_dim_size--piece_size
    int64_t acc_strides = piece_size;
    for (int it = 0; it < indices_last_dim_size; ++it) {
        input_strides[indices_last_dim_size - 1 - it] = acc_strides;
        acc_strides *= input_shape->GetDim(indices_last_dim_size - 1 - it);
    }
    for (int it = 0; it < num_input_dim; ++it)
        input_dims[it] = input_shape->GetDim(it);
    hipMemcpyAsync(input_strides_gpu, input_strides.data(), sizeof(int64_t) * indices_last_dim_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(input_dims_gpu, input_dims.data(), sizeof(int64_t) * num_input_dim, hipMemcpyHostToDevice, stream);
    int cal_offset_grid = (num_pieces + block_size - 1) / block_size;
    switch (ppl::common::GetSizeOfDataType(indices_shape->GetDataType())) {
        case sizeof(int32_t): {
            ppl_cukernel_scatter_nd_offset<<<cal_offset_grid, block_size, 0, stream>>>(num_pieces, input_dims_gpu, input_strides_gpu, indices_last_dim_size, (const int32_t*)indices, piece_offsets);
            break;
        }
        case sizeof(int64_t): {
            ppl_cukernel_scatter_nd_offset<<<cal_offset_grid, block_size, 0, stream>>>(num_pieces, input_dims_gpu, input_strides_gpu, indices_last_dim_size, (const int64_t*)indices, piece_offsets);
            break;
        }
        default:
            return ppl::common::RC_UNSUPPORTED;
    }

    // step2: begiin scatter elements
    int64_t num_elems     = updates_shape->GetElementsIncludingPadding();
    int scatter_grid_size = (num_elems + block_size - 1) / block_size;
    DivModFast piece_size_fast(piece_size);

#define SWITCH_CASE(TYPE)                                                                                                                                      \
    case sizeof(TYPE): {                                                                                                                                       \
        ppl_cukernel_scatter_nd<<<scatter_grid_size, block_size, 0, stream>>>(num_elems, piece_size_fast, piece_offsets, (const TYPE*)updates, (TYPE*)output); \
        return ppl::common::RC_SUCCESS;                                                                                                                        \
    }

    switch (ppl::common::GetSizeOfDataType(input_shape->GetDataType())) {
        SWITCH_CASE(int8_t);
        SWITCH_CASE(int16_t);
        SWITCH_CASE(int32_t);
        SWITCH_CASE(int64_t);
        default:
            return ppl::common::RC_UNSUPPORTED;
    }

#undef SWITCH_CASE
}
