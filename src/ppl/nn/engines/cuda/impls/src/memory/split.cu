#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "cudakernel/memory/split.h"
#include "cudakernel/memory/slice.h"
#include "cudakernel/common/divmod_fast.h"
#include "cudakernel/common/common.h"
#include "ppl/nn/common/tensor_shape.h"
#include "ppl/common/retcode.h"
#include <hip/hip_runtime.h>

#define NHWC8_ALIGNED_AXIS (8)

template <typename T1, typename T2>
__global__ void __launch_bounds__(256) ppl_cukernel_split_nhwc_two_inputs(
    int64_t num_elems,
    int inner_dims,
    int pad_inner_dims,
    int axis_width0,
    int pad_axis_width0,
    int axis_width1,
    int pad_axis_width1,
    T1* input,
    T2* output0,
    T2* output1)
{
    for (int64_t i = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;
         i < num_elems;
         i += (int64_t)blockDim.x * gridDim.x) {
        int inner_idx    = i % inner_dims;
        int outer_idx    = i / inner_dims;
        int input_offset = outer_idx * pad_inner_dims + inner_idx;
        if (inner_idx >= axis_width0) {
            int output_offset      = outer_idx * pad_axis_width1 + (inner_idx - axis_width0);
            output1[output_offset] = input[input_offset];
        } else {
            int output_offset      = outer_idx * pad_axis_width0 + inner_idx;
            output0[output_offset] = input[input_offset];
        }
    }
}

template <typename T>
__global__ void __launch_bounds__(256) ppl_cukernel_split_ndarray(
    int64_t num_elems,
    DivModFast inner_dims_fast,
    int in_split_axis_size,
    DivModFast out_split_axis_size_fast,
    int offset_split_axis,
    const T* input,
    T* output)
{
    for (int64_t i = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;
         i < num_elems;
         i += (int64_t)blockDim.x * gridDim.x) {
        int inner_idx, outer_split_idx;
        inner_dims_fast.divmod(i, outer_split_idx, inner_idx);
        int split_idx, outer_idx;
        out_split_axis_size_fast.divmod(outer_split_idx, outer_idx, split_idx);
        int inner_dims   = inner_dims_fast.d_;
        int input_offset = outer_idx * in_split_axis_size * inner_dims +
                           (split_idx + offset_split_axis) * inner_dims + inner_idx;
        output[i] = input[input_offset];
    }
}

ppl::common::RetCode PPLCUDASplitForwardImp(
    hipStream_t stream,
    int split_axis,
    const ppl::nn::TensorShape* input_shape,
    const void* input,
    int num_outputs,
    const int64_t* out_dims[],
    void* outputs[])
{
    int64_t num_byte_elem = ppl::common::GetSizeOfDataType(input_shape->GetDataType());
    if (input_shape->GetDataFormat() == ppl::common::DATAFORMAT_NDARRAY) {
        int num_dims              = input_shape->GetDimCount();
        int64_t split_size        = 1;
        int64_t split_count       = 1;
        int64_t offset_split_axis = 0;
        int64_t split_axis_length = input_shape->GetDim(split_axis);
        for (int i = 0; i < split_axis; i++)
            split_count *= input_shape->GetDim(i);
        for (int i = split_axis + 1; i < num_dims; i++)
            split_size *= input_shape->GetDim(i);

#define SWITCH_CASE(TYPE)                                                                                                                                                                         \
    case sizeof(TYPE): {                                                                                                                                                                          \
        for (int i = 0; i < num_outputs; i++) {                                                                                                                                                   \
            TYPE* out_ptr             = static_cast<TYPE*>(outputs[i]);                                                                                                                           \
            const TYPE* input_ptr     = static_cast<const TYPE*>(input);                                                                                                                          \
            int out_split_axis_length = out_dims[i][split_axis];                                                                                                                                  \
            int split_size_with_axis  = out_split_axis_length * split_size;                                                                                                                       \
            int memcpy_threshold      = 64;                                                                                                                                                       \
            if (split_size_with_axis < memcpy_threshold) {                                                                                                                                        \
                int block_size = 128;                                                                                                                                                             \
                int out_elems  = split_size_with_axis * split_count;                                                                                                                              \
                int grid_size  = (out_elems + block_size - 1) / block_size;                                                                                                                       \
                DivModFast split_size_fast(split_size);                                                                                                                                           \
                DivModFast out_split_axis_size_fast(out_split_axis_length);                                                                                                                       \
                ppl_cukernel_split_ndarray<<<grid_size, block_size, 0, stream>>>(out_elems, split_size_fast, split_axis_length, out_split_axis_size_fast, offset_split_axis, input_ptr, out_ptr); \
            } else {                                                                                                                                                                              \
                for (int n = 0; n < split_count; n++) {                                                                                                                                           \
                    int64_t out_offset = n * split_size_with_axis;                                                                                                                                \
                    int64_t in_offset  = (n * split_axis_length + offset_split_axis) * split_size;                                                                                                \
                    hipMemcpyAsync(out_ptr + out_offset, input_ptr + in_offset, split_size_with_axis * num_byte_elem, hipMemcpyDeviceToDevice, stream);                                         \
                }                                                                                                                                                                                 \
            }                                                                                                                                                                                     \
            offset_split_axis += out_split_axis_length;                                                                                                                                           \
        }                                                                                                                                                                                         \
        return ppl::common::RC_SUCCESS;                                                                                                                                                           \
    }

        switch (num_byte_elem) {
            SWITCH_CASE(int8_t);
            SWITCH_CASE(int16_t);
            SWITCH_CASE(int32_t);
            SWITCH_CASE(int64_t);
            default:
                return ppl::common::RC_UNSUPPORTED;
        }

#undef SWITCH_CASE
    } else if (input_shape->GetDataFormat() == ppl::common::DATAFORMAT_NHWC8 ||
               input_shape->GetDataFormat() == ppl::common::DATAFORMAT_NHWC16) {
        int num_dims = input_shape->GetDimCount();
        if (num_dims < 2)
            return ppl::common::RC_UNSUPPORTED;
        int input_elems = input_shape->GetElementsExcludingPadding();
        if (num_outputs == 2 && split_axis == 1) {
            int align_size = NHWC8_ALIGNED_AXIS;
            if (input_shape->GetDataFormat() == ppl::common::DATAFORMAT_NHWC16) {
                align_size = NHWC8_ALIGNED_AXIS * 2;
            }
#define SWITCH_CASE(TYPE)                                                                            \
    case sizeof(TYPE): {                                                                             \
        int block_size      = 256;                                                                   \
        int grid_size       = (input_elems + block_size - 1) / block_size;                           \
        int axis_width0     = out_dims[0][split_axis];                                               \
        int pad_axis_width0 = Align(axis_width0, align_size);                                \
        int axis_width1     = out_dims[1][split_axis];                                               \
        int pad_axis_width1 = Align(axis_width1, align_size);                                \
        int inner_dims      = axis_width0 + axis_width1;                                             \
        int pad_inner_dims  = Align(inner_dims, align_size);                                 \
        ppl_cukernel_split_nhwc_two_inputs<<<grid_size, block_size, 0, stream>>>(input_elems,        \
                                                                                 inner_dims,         \
                                                                                 pad_inner_dims,     \
                                                                                 axis_width0,        \
                                                                                 pad_axis_width0,    \
                                                                                 axis_width1,        \
                                                                                 pad_axis_width1,    \
                                                                                 (const TYPE*)input, \
                                                                                 (TYPE*)outputs[0],  \
                                                                                 (TYPE*)outputs[1]); \
        return ppl::common::RC_SUCCESS;                                                              \
    }

            switch (num_byte_elem) {
                SWITCH_CASE(int8_t);
                SWITCH_CASE(int16_t);
                SWITCH_CASE(int32_t);
                SWITCH_CASE(int64_t);
                default:
                    return ppl::common::RC_UNSUPPORTED;
            }
#undef SWITCH_CASE
        }

        ppl::nn::TensorShape output_shape(*input_shape);

        SliceKernelParam param;
        param.axes_num  = 1;
        param.starts[0] = 0;
        param.ends[0]   = input_shape->GetDim(split_axis);
        param.axes[0]   = split_axis;
        param.steps[0]  = 1;

        int64_t offset_split_axis = 0;
        for (int i = 0; i < num_outputs; i++) {
            output_shape.Reshape(out_dims[i], num_dims);
            output_shape.CalcPadding();
            param.starts[0]           = offset_split_axis;
            int out_split_axis_length = out_dims[i][split_axis];
            offset_split_axis += out_split_axis_length;
            param.ends[0] = offset_split_axis;
            PPLCUDASliceForwardImp(stream, param, input_shape, input, &output_shape, outputs[i]);
        }
        return ppl::common::RC_SUCCESS;
    } else {
        return ppl::common::RC_UNSUPPORTED;
    }
}