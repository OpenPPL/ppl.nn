#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "cudakernel/memory/pad.h"
#include "cudakernel/common/divmod_fast.h"
#include "cudakernel/common/memory_utils.h"
#include "cudakernel/common/common.h"
#include "ppl/common/tensor_shape.h"
#include "ppl/common/retcode.h"
#include <hip/hip_fp16.h>

template <int MODE>
__device__ int pad_calc_in_idx(int out_idx, int64_t start_pad_val, int64_t input_dim, bool& use_pad_value) {
    int res = 0;
    if (out_idx < start_pad_val || out_idx >= start_pad_val + input_dim)
        use_pad_value = true;
    else
        res = out_idx - start_pad_val;
    return res;
}

// PadKernelParam::PAD_MODE_REFLECT --> 1
template <>
__device__ int pad_calc_in_idx<PadKernelParam::PAD_MODE_REFLECT>(int out_idx, int64_t start_pad_val, int64_t input_dim, bool& use_pad_value) {
    int res = 0;
    if (out_idx < start_pad_val) {
        res = start_pad_val - out_idx;
    } else if (out_idx >= start_pad_val + input_dim) {
        res = input_dim - 2 - (out_idx - (start_pad_val + input_dim));
    } else {
        res = out_idx - start_pad_val;
    }
    return res;
}

// PadKernelParam::PAD_MODE_EDGE --> 1
template <>
__device__ int pad_calc_in_idx<PadKernelParam::PAD_MODE_EDGE>(int out_idx, int64_t start_pad_val, int64_t input_dim, bool& use_pad_value) {
    int res = 0;
    if (out_idx < start_pad_val) {
        res = 0;
    } else if (out_idx >= start_pad_val + input_dim) {
        res = input_dim - 1;
    } else {
        res = out_idx - start_pad_val;
    }
    return res;
}

template <typename T, int MODE>
__global__ void ppl_cukernel_pad(
    int64_t num_elems,
    int num_dims,
    PadKernelParam param,
    GArray<int64_t> input_dims,
    GArray<int64_t> input_strides,
    const T* input,
    const int64_t* pads,
    GArray<DivModFast> output_strides_fast,
    T* output)
{
    int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_elems)
        return;
    bool use_pad_value   = false;
    int64_t input_offset = 0;
    int out_idx, remain = index;
    for (int it = 0; (it < num_dims) && !use_pad_value; ++it) {
        output_strides_fast[it].divmod(remain, out_idx, remain);
        int64_t start_pad_val = pads[it];
        int in_idx            = 0;
        in_idx = pad_calc_in_idx<MODE>(out_idx, start_pad_val, input_dims[it], use_pad_value);
        input_offset += in_idx * input_strides[it];
    }
    output[index] = use_pad_value ? (T)param.constant_value : input[input_offset];
}
bool isFastPadSupported(const std::vector<int32_t>& pads, int32_t num_dims) {
    if (num_dims < 3) return false;
    int32_t diff_cnt = num_dims - 2;
    for (int32_t i = 0; i < diff_cnt; ++i) {
        if (pads[i] != 0) return false; // start
        if (pads[num_dims + i] != 0) return false; //end
    }
    if (pads[num_dims - 1] != 0) return false;
    if (pads[num_dims - 2] != 0) return false;
    return true;
}

template <typename T>
__global__ void ppl_cukernel_pad_fast(const T* input, int src_height, int src_width,
    T* output, int dst_height, int dst_width) {
    int dst_hgt = blockIdx.y * blockDim.y + threadIdx.y;
    int dst_wdt = blockIdx.x * blockDim.x + threadIdx.x;
    if (dst_hgt >= dst_height || dst_wdt >= dst_width) return;
    int b_idx = blockIdx.z;
    int dst_idx = b_idx * dst_height * dst_width + dst_hgt * dst_width + dst_wdt;
    if (dst_hgt >= src_height || dst_wdt >= src_width) {
        output[dst_idx] = T(0);
    } else {
        int src_idx = b_idx * src_height * src_width + dst_hgt * src_width + dst_wdt;
        output[dst_idx] = input[src_idx];
    }
}

// last 2-dim padded
bool isFastPadSupported2(const std::vector<int32_t>& pads, int32_t num_dims) {
    if (num_dims < 3) return false;
    int32_t diff_cnt = num_dims - 2;
    for (int32_t i = 0; i < diff_cnt; ++i) {
        if (pads[i] != 0) return false; // start
        if (pads[num_dims + i] != 0) return false; //end
    }
    return true;
}

template <typename T, int MODE>
__global__ void ppl_cukernel_pad_fast2(const T* input, int src_height, int src_width,
    T* output, int dst_height, int dst_width, int num_dims, const int64_t* pads, PadKernelParam param) {
    int dst_hgt = blockIdx.y * blockDim.y + threadIdx.y;
    int dst_wdt = blockIdx.x * blockDim.x + threadIdx.x;
    if (dst_hgt >= dst_height || dst_wdt >= dst_width) return;
    int b_idx = blockIdx.z;
    int dst_idx = b_idx * dst_height * dst_width + dst_hgt * dst_width + dst_wdt;
    bool use_pad_value = false;
    int in_hgt = pad_calc_in_idx<MODE>(dst_hgt, pads[num_dims - 2], src_height, use_pad_value);
    int in_wdt = pad_calc_in_idx<MODE>(dst_wdt, pads[num_dims - 1], src_width, use_pad_value);
    int src_idx = b_idx * src_height * src_width + in_hgt * src_width + in_wdt;
    output[dst_idx] = use_pad_value ? (T)param.constant_value : input[src_idx];
}

ppl::common::RetCode PPLCUDAPadForwardImp(
    hipStream_t stream,
    PadKernelParam param,
    ppl::common::TensorShape* input_shape,
    const void* input,
    const int64_t* pads,
    ppl::common::TensorShape* output_shape,
    void* output)
{
    int num_dims       = output_shape->GetDimCount();
    if (isFastPadSupported(param.pads, num_dims)) {
        int batch = input_shape->CalcElementsToDimensionExcludingPadding(num_dims - 2);
        int dst_height = output_shape->GetDim(num_dims - 2);
        int dst_width  = output_shape->GetDim(num_dims - 1);
        int src_height = input_shape->GetDim(num_dims - 2);
        int src_width  = input_shape->GetDim(num_dims - 1);
        dim3 block_size(16, 16, 1);
        dim3 grid_size(DivUp(dst_width, 16), DivUp(dst_height, 16), batch);
        switch (input_shape->GetDataType()) {
            case ppl::common::DATATYPE_INT8: {
                ppl_cukernel_pad_fast<<<grid_size, block_size, 0, stream>>>(
                    (const int8_t*)input, src_height, src_width, (int8_t*)output, dst_height, dst_width);
                return ppl::common::RC_SUCCESS;
            }
            case ppl::common::DATATYPE_FLOAT16: {
                ppl_cukernel_pad_fast<<<grid_size, block_size, 0, stream>>>(
                    (const half*)input, src_height, src_width, (half*)output, dst_height, dst_width);
                return ppl::common::RC_SUCCESS;
            }
            case ppl::common::DATATYPE_FLOAT32: {
                ppl_cukernel_pad_fast<<<grid_size, block_size, 0, stream>>>(
                    (const float*)input, src_height, src_width, (float*)output, dst_height, dst_width);
                return ppl::common::RC_SUCCESS;
            }
            default:
                return ppl::common::RC_UNSUPPORTED;
        }
    } else if (isFastPadSupported2(param.pads, num_dims)) {
        int batch = input_shape->CalcElementsToDimensionExcludingPadding(num_dims - 2);
        int dst_height = output_shape->GetDim(num_dims - 2);
        int dst_width  = output_shape->GetDim(num_dims - 1);
        int src_height = input_shape->GetDim(num_dims - 2);
        int src_width  = input_shape->GetDim(num_dims - 1);
        dim3 block_size(16, 16, 1);
        dim3 grid_size(DivUp(dst_width, 16), DivUp(dst_height, 16), batch);
#define PAD_EXEC_FAST2(TYPE, MODE) \
    ppl_cukernel_pad_fast2<TYPE, MODE><<<grid_size, block_size, 0, stream>>>( \
                    (const TYPE*)input, src_height, src_width, (TYPE*)output, dst_height, dst_width, \
                    num_dims, pads, param); \
    break;

        switch (input_shape->GetDataType()) {
            case ppl::common::DATATYPE_INT8: {
                switch(param.mode) {
                    case PadKernelParam::PAD_MODE_CONSTANT:
                        PAD_EXEC_FAST2(int8_t, PadKernelParam::PAD_MODE_CONSTANT)
                    case PadKernelParam::PAD_MODE_REFLECT:
                        PAD_EXEC_FAST2(int8_t, PadKernelParam::PAD_MODE_REFLECT)
                    case PadKernelParam::PAD_MODE_EDGE:
                        PAD_EXEC_FAST2(int8_t, PadKernelParam::PAD_MODE_EDGE)
                }
                return ppl::common::RC_SUCCESS;
            }
            case ppl::common::DATATYPE_FLOAT16: {
                switch(param.mode) {
                    case PadKernelParam::PAD_MODE_CONSTANT:
                        PAD_EXEC_FAST2(half, PadKernelParam::PAD_MODE_CONSTANT)
                    case PadKernelParam::PAD_MODE_REFLECT:
                        PAD_EXEC_FAST2(half, PadKernelParam::PAD_MODE_REFLECT)
                    case PadKernelParam::PAD_MODE_EDGE:
                        PAD_EXEC_FAST2(half, PadKernelParam::PAD_MODE_EDGE)
                }
                return ppl::common::RC_SUCCESS;
            }
            case ppl::common::DATATYPE_FLOAT32: {
                switch(param.mode) {
                    case PadKernelParam::PAD_MODE_CONSTANT:
                        PAD_EXEC_FAST2(float, PadKernelParam::PAD_MODE_CONSTANT)
                    case PadKernelParam::PAD_MODE_REFLECT:
                        PAD_EXEC_FAST2(float, PadKernelParam::PAD_MODE_REFLECT)
                    case PadKernelParam::PAD_MODE_EDGE:
                        PAD_EXEC_FAST2(float, PadKernelParam::PAD_MODE_EDGE)
                }
                return ppl::common::RC_SUCCESS;
            }
            default:
                return ppl::common::RC_UNSUPPORTED;
            }
    }
    int block_size     = 256;
    uint64_t num_elems = output_shape->CalcElementsIncludingPadding();
    int grid_size      = (num_elems + block_size - 1) / block_size;
    GArray<int64_t> input_dims(num_dims);
    GArray<int64_t> input_strides(num_dims);
    GArray<DivModFast> output_strides_fast(num_dims);
    int64_t acc_output_stride = 1;
    int64_t acc_input_stride  = 1;
    for (int it = num_dims - 1; it >= 0; --it) {
        input_dims[it]          = input_shape->GetDim(it);
        input_strides[it]       = acc_input_stride;
        output_strides_fast[it] = DivModFast(acc_output_stride);
        acc_input_stride *= input_shape->GetDim(it);
        acc_output_stride *= output_shape->GetDim(it);
    }

#define PAD_EXEC(TYPE, MODE) \
    ppl_cukernel_pad<TYPE, MODE><<<grid_size, block_size, 0, stream>>>( \
                num_elems, num_dims, param, input_dims, input_strides, (const TYPE*)input, pads, output_strides_fast, (TYPE*)output); \
    break;

    switch (input_shape->GetDataType()) {
        case ppl::common::DATATYPE_INT8: {
            switch(param.mode) {
                case PadKernelParam::PAD_MODE_CONSTANT:
                    PAD_EXEC(int8_t, PadKernelParam::PAD_MODE_CONSTANT)
                case PadKernelParam::PAD_MODE_REFLECT:
                    PAD_EXEC(int8_t, PadKernelParam::PAD_MODE_REFLECT)
                case PadKernelParam::PAD_MODE_EDGE:
                    PAD_EXEC(int8_t, PadKernelParam::PAD_MODE_EDGE)
            }
            return ppl::common::RC_SUCCESS;
        }
        case ppl::common::DATATYPE_FLOAT16: {
            switch(param.mode) {
                case PadKernelParam::PAD_MODE_CONSTANT:
                    PAD_EXEC(half, PadKernelParam::PAD_MODE_CONSTANT)
                case PadKernelParam::PAD_MODE_REFLECT:
                    PAD_EXEC(half, PadKernelParam::PAD_MODE_REFLECT)
                case PadKernelParam::PAD_MODE_EDGE:
                    PAD_EXEC(half, PadKernelParam::PAD_MODE_EDGE)
            }
            return ppl::common::RC_SUCCESS;
        }
        case ppl::common::DATATYPE_FLOAT32: {
            switch(param.mode) {
                case PadKernelParam::PAD_MODE_CONSTANT:
                    PAD_EXEC(float, PadKernelParam::PAD_MODE_CONSTANT)
                case PadKernelParam::PAD_MODE_REFLECT:
                    PAD_EXEC(float, PadKernelParam::PAD_MODE_REFLECT)
                case PadKernelParam::PAD_MODE_EDGE:
                    PAD_EXEC(float, PadKernelParam::PAD_MODE_EDGE)
            }
            return ppl::common::RC_SUCCESS;
        }
        default:
            return ppl::common::RC_UNSUPPORTED;
    }
}
