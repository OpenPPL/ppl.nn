#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "cudakernel/memory/gather.h"
#include "cudakernel/common/divmod_fast.h"
#include "ppl/nn/common/tensor_shape.h"
#include "ppl/common/retcode.h"
#include <hip/hip_runtime.h>
#include <memory>

__host__ __device__ __inline__ int get_indices_val(
    int indices_element_size,
    int offset,
    const void* indices)
{
    int res = 0;
    switch (indices_element_size) {
        case sizeof(int32_t):
            res = static_cast<const int32_t*>(indices)[offset];
            break;
        case sizeof(int64_t):
            res = static_cast<const int64_t*>(indices)[offset];
            break;
        default:
            break;
    }
    return res;
}

template <typename T>
__global__ void ppl_cukernel_gather(
    int64_t num_elems,
    DivModFast output_outer_block_fast,
    int input_axis_size,
    DivModFast output_inner_block_fast,
    const T* input,
    T* output,
    int indices_element_size,
    const void* indices)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_elems)
        return;
    int outer_idx, block_offset;
    output_outer_block_fast.divmod(index, outer_idx, block_offset);
    int indices_offset, inner_idx;
    output_inner_block_fast.divmod(block_offset, indices_offset, inner_idx);
    int64_t indices_idx = get_indices_val(indices_element_size, indices_offset, indices);
    // -d means distance from last dimension
    indices_idx         = indices_idx < 0 ? indices_idx + input_axis_size : indices_idx;
    if (indices_idx < 0 || indices_idx >= input_axis_size) {
        output[index] = 0;
        return;
    }
    int64_t input_idx = (outer_idx * input_axis_size + indices_idx) *
                            output_inner_block_fast.d_ +
                        inner_idx;
    output[index] = input[input_idx];
}

ppl::common::RetCode PPLCUDAGatherForwardImp(
    hipStream_t stream,
    const ppl::nn::TensorShape* input_shape,
    const void* input,
    const ppl::nn::TensorShape* indices_shape,
    const void* indices,
    const ppl::nn::TensorShape* output_shape,
    void* output,
    int axis)
{
    int indices_element_size = ppl::common::GetSizeOfDataType(indices_shape->GetDataType());
    // special case, need further evaluement (performance is not usually better)
    if (axis == 0 && indices_shape->GetDimCount() == 1 && indices_shape->GetDim(0) == 1) {
        int indices_data_size = indices_shape->GetBytesIncludingPadding();
        std::unique_ptr<char[]> indices_data(new char[indices_data_size]);
        hipMemcpy(indices_data.get(), indices, indices_data_size, hipMemcpyDeviceToHost);
        int inner_size   = input_shape->GetBytesIncludingPadding() / input_shape->GetDim(0);
        int input_offset = get_indices_val(indices_element_size, 0, indices_data.get());
        hipMemcpy(output, static_cast<const char*>(input) + input_offset * inner_size, output_shape->GetBytesIncludingPadding(), hipMemcpyDeviceToDevice);
        return ppl::common::RC_SUCCESS;
    }
    int64_t num_elems      = output_shape->GetElementsIncludingPadding();
    int block_size         = 256;
    int grid_size          = (num_elems + block_size - 1) / block_size;
    // output dimension can be partitioned as outer--indices--inner. (before axis, axis, after axis)
    int output_inner_block = input_shape->GetElementsFromDimensionIncludingPadding(axis + 1);
    int input_axis_size    = input_shape->GetDim(axis);
    int indices_block_size = indices_shape->GetElementsIncludingPadding();
    int output_outer_block = indices_block_size * output_inner_block;

    DivModFast output_outer_block_fast(output_outer_block);
    DivModFast output_inner_block_fast(output_inner_block);

#define SWITCH_CASE(TYPE)                                                                                                                                                                                                       \
    case sizeof(TYPE): {                                                                                                                                                                                                        \
        ppl_cukernel_gather<<<grid_size, block_size, 0, stream>>>(num_elems, output_outer_block_fast, input_axis_size, output_inner_block_fast, (const TYPE*)input, (TYPE*)output, indices_element_size, (const void*)indices); \
        return ppl::common::RC_SUCCESS;                                                                                                                                                                                         \
    }

    switch (ppl::common::GetSizeOfDataType(input_shape->GetDataType())) {
        SWITCH_CASE(int8_t);
        SWITCH_CASE(int16_t);
        SWITCH_CASE(int32_t);
        SWITCH_CASE(int64_t);
        default:
            return ppl::common::RC_UNSUPPORTED;
    }

#undef SWITCH_CASE
}
