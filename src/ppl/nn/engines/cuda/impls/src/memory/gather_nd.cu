#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "cudakernel/memory/gather_nd.h"
#include "cudakernel/common/divmod_fast.h"
#include "ppl/nn/common/tensor_shape.h"
#include "ppl/common/retcode.h"
#include "ppl/common/types.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include <vector>

template <typename T>
__global__ void ppl_cukernel_gather_nd(
    int64_t num_elems,
    DivModFast piece_size_fast,
    int64_t* piece_offsets,
    const T* input,
    T* output)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_elems)
        return;
    int piece_idx, offset;
    piece_size_fast.divmod(index, piece_idx, offset);
    int64_t base_offset = piece_offsets[piece_idx];
    output[index]       = input[base_offset + offset];
}

template <typename IndexT>
__global__ void ppl_cukernel_gather_nd_offset(
    int64_t num_pieces,
    DivModFast num_pieces_per_batch_fast,
    int batch_dim,
    int64_t* input_dims_gpu,
    int input_batch_stride,
    int64_t* input_strides_gpu,
    int indices_last_dim_size,
    const IndexT* indices_data,
    int64_t* piece_offsets)
{
    int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_pieces)
        return;
    // batch offset
    int batch_idx             = num_pieces_per_batch_fast.div(index);
    int64_t batch_offset      = batch_idx * input_batch_stride;
    // inner offset
    const IndexT* indices_ptr = indices_data + index * indices_last_dim_size;
    int64_t rel_offset        = 0;
    for (int it = 0; it < indices_last_dim_size; ++it) {
        IndexT cor_val = indices_ptr[it];
        if (cor_val < 0)
            cor_val += input_dims_gpu[batch_dim + it];
        assert(cor_val >= 0 && cor_val < input_dims_gpu[batch_dim + it]);
        rel_offset += cor_val * input_strides_gpu[it];
    }
    piece_offsets[index] = batch_offset + rel_offset;
}

int64_t pplGatherNDGetTempBufferSize(
    const ppl::nn::TensorShape* input_shape,
    const void* input,
    const ppl::nn::TensorShape* indices_shape,
    const void* indices)
{
    int num_input_dim   = input_shape->GetDimCount();
    int num_indices_dim = indices_shape->GetDimCount();
    int num_pieces      = indices_shape->GetElementsToDimensionIncludingPadding(num_indices_dim - 1);
    // pieces offsets and input strides and input_dims
    int64_t total_size  = (num_pieces + 2 * num_input_dim) * sizeof(int64_t);
    return total_size;
}

ppl::common::RetCode PPLCUDAGatherNDForwardImp(
    hipStream_t stream,
    const ppl::nn::TensorShape* input_shape,
    const void* input,
    const ppl::nn::TensorShape* indices_shape,
    const void* indices,
    const ppl::nn::TensorShape* output_shape,
    void* output,
    void* temp_buffer,
    int batch_dim)
{
    int num_batches           = input_shape->GetElementsToDimensionIncludingPadding(batch_dim);
    int input_batch_stride    = input_shape->GetElementsFromDimensionIncludingPadding(batch_dim);
    int num_indices_dim       = indices_shape->GetDimCount();
    int num_input_dim         = input_shape->GetDimCount();
    int indices_last_dim_size = indices_shape->GetDim(num_indices_dim - 1);
    int num_pieces            = indices_shape->GetElementsToDimensionIncludingPadding(num_indices_dim - 1);
    DivModFast num_pieces_per_batch_fast(num_pieces / num_batches);
    int piece_size = input_shape->GetElementsFromDimensionIncludingPadding(
        batch_dim + indices_last_dim_size);
    int block_size             = 256;
    // step 1: calcalute each piece's offset first
    int64_t* piece_offsets     = static_cast<int64_t*>(temp_buffer);
    int64_t* input_strides_gpu = piece_offsets + num_pieces;
    int64_t* input_dims_gpu    = input_strides_gpu + num_input_dim;
    std::vector<int64_t> input_strides(indices_last_dim_size);
    std::vector<int64_t> input_dims(num_input_dim);
    // dimension is partitioned as batch--indices_last_dim_size--piece_size
    int64_t acc_strides = piece_size;
    for (int it = 0; it < indices_last_dim_size; ++it) {
        input_strides[indices_last_dim_size - 1 - it] = acc_strides;
        acc_strides *= input_shape->GetDim(batch_dim + indices_last_dim_size - 1 - it);
    }
    for (int it = 0; it < num_input_dim; ++it)
        input_dims[it] = input_shape->GetDim(it);
    hipMemcpyAsync(input_strides_gpu, input_strides.data(), sizeof(int64_t) * indices_last_dim_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(input_dims_gpu, input_dims.data(), sizeof(int64_t) * num_input_dim, hipMemcpyHostToDevice, stream);
    int cal_offset_grid = (num_pieces + block_size - 1) / block_size;
    switch (ppl::common::GetSizeOfDataType(indices_shape->GetDataType())) {
        case sizeof(int32_t): {
            ppl_cukernel_gather_nd_offset<<<cal_offset_grid, block_size, 0, stream>>>(num_pieces, num_pieces_per_batch_fast, batch_dim, input_dims_gpu, input_batch_stride, input_strides_gpu, indices_last_dim_size, (const int32_t*)indices, piece_offsets);
            break;
        }
        case sizeof(int64_t): {
            ppl_cukernel_gather_nd_offset<<<cal_offset_grid, block_size, 0, stream>>>(num_pieces, num_pieces_per_batch_fast, batch_dim, input_dims_gpu, input_batch_stride, input_strides_gpu, indices_last_dim_size, (const int64_t*)indices, piece_offsets);
            break;
        }
        default:
            return ppl::common::RC_UNSUPPORTED;
    }

    // step2: begiin gather elements
    int64_t num_elems    = output_shape->GetElementsIncludingPadding();
    int gather_grid_size = (num_elems + block_size - 1) / block_size;
    DivModFast piece_size_fast(piece_size);

#define SWITCH_CASE(TYPE)                                                                                                                                  \
    case sizeof(TYPE): {                                                                                                                                   \
        ppl_cukernel_gather_nd<<<gather_grid_size, block_size, 0, stream>>>(num_elems, piece_size_fast, piece_offsets, (const TYPE*)input, (TYPE*)output); \
        return ppl::common::RC_SUCCESS;                                                                                                                    \
    }

    switch (ppl::common::GetSizeOfDataType(input_shape->GetDataType())) {
        SWITCH_CASE(int8_t);
        SWITCH_CASE(int16_t);
        SWITCH_CASE(int32_t);
        SWITCH_CASE(int64_t);
        default:
            return ppl::common::RC_UNSUPPORTED;
    }

#undef SWITCH_CASE
}
