// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "cudakernel/memory/squeeze.h"
#include "ppl/common/tensor_shape.h"
#include "ppl/common/retcode.h"
#include "ppl/common/types.h"
#include <hip/hip_runtime.h>

ppl::common::RetCode PPLCUDASqueezeForwardImp(
    hipStream_t stream,
    const ppl::common::TensorShape* input_shape,
    const void* input,
    const ppl::common::TensorShape* output_shape,
    void* output)
{
    int64_t num_elems_output = output_shape->CalcElementsIncludingPadding();
    hipMemcpyAsync(output, input, ppl::common::GetSizeOfDataType(input_shape->GetDataType()) * num_elems_output, hipMemcpyDeviceToDevice, stream);
    return ppl::common::RC_SUCCESS;
}