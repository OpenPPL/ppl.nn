// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "conv_depthwise_kernel.h"
#include "ppl/nn/common/tensor_shape.h"
#include "cudakernel/nn/conv_fuse_type.h"

#include "depthwise_info.h"

#include <float.h>
#include <vector>

#if defined(_WIN32) || defined(_WIN64)
#define __FLT_MAX__ 3.402823466e+38F
#endif

static std::vector<depthwise_kernel_info> func_vec;

void PPLCUDADepthwiseConvertFilter(
    hipStream_t& stream,
    void* filter,
    void* cvt_filter,
    struct conv_param_t &conv_param,
    ppl::common::datatype_t type)
{
    int in_height  = conv_param.num_chl;
    int in_width   = conv_param.flt_height * conv_param.flt_width;
    int out_width  = conv_param.num_chl_pad;
    int out_height = in_width;

    int block_size = 32;
    int num_bx     = DivUp(out_height, block_size);
    int num_by     = DivUp(out_width, block_size);
    dim3 dim_grid(num_bx, num_by, 1);
    dim3 dim_block(block_size, 1, 1);

    if(type == ppl::common::DATATYPE_FLOAT16) {
        ppl_cukernel_matrix_transpose<half><<<dim_grid, dim_block, 0, stream>>>(
        (const half*)filter, (half*)cvt_filter, in_height, in_width, out_height, out_width);
    } else if(type == ppl::common::DATATYPE_FLOAT32) {
        ppl_cukernel_matrix_transpose<float><<<dim_grid, dim_block, 0, stream>>>(
        (const float*)filter, (float*)cvt_filter, in_height, in_width, out_height, out_width);
    } else if(type == ppl::common::DATATYPE_INT8) {
        ppl_cukernel_matrix_transpose_int8<<<dim_grid, dim_block, 0, stream>>>(
        (const int8_t*)filter, (int8_t*)cvt_filter, in_height, in_width, out_height, out_width);
    }
    
}
int PPLCUDADepthwiseSelectKernel(
    hipStream_t& stream,
    void* input,
    void* filter,
    void* bias,
    int times,
	struct conv_param_t &conv_param, 
	struct fuse_param_t &fuse_param,
    void* output,
    ppl::common::datatype_t type,
    float pic_scale,
    float* flt_scale,
    float out_scale)
{
    GETPARAM
    if(func_vec.empty())  InitKernelList(func_vec, type);
    int kernel_id = 0;
    float min_time = FLT_MAX;
    float elapsed;
    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);
    for (uint32_t id = 0; id < func_vec.size(); id++) {
        if (!CanSupport(func_vec[id], conv_param))
            continue;
        hipEventRecord(begin, stream);
        for (int i = 0; i < 10; i++) {
            int tile_height, tile_width, elems;
            GenConfigure(func_vec[id], conv_param, &tile_height, &tile_width, &elems);
            dim3 dim_block(BLOCK_SIZE,1,1), dim_grid(DivUp(elems,BLOCK_SIZE), 1, 1);
            DivModFast padc_fast(paddingc);
            DivModFast hw_fast(tile_height * tile_width);
            DivModFast width_fast(tile_width);
            if(type == ppl::common::DATATYPE_FLOAT16) {
                func_vec[id].kernel_ptr_half<<<dim_grid, dim_block, 0, stream>>>((const half*)input, (const half*)filter, (const half*)bias, 
                padc_fast, hw_fast, width_fast,
                in_height, in_width, kernel_h, kernel_w, pad_h, pad_w, stride_h, stride_w, hole_h, hole_w,
                tile_height, tile_width, channels, paddingc, out_height, out_width, 
                in_batch_stride, in_height_stride, in_width_stride, elems, (half*)output, fuse_param);
            } else if(type == ppl::common::DATATYPE_FLOAT32) {
                func_vec[id].kernel_ptr_float<<<dim_grid, dim_block, 0, stream>>>((const float*)input, (const float*)filter, (const float*)bias, 
                padc_fast, hw_fast, width_fast,
                in_height, in_width, kernel_h, kernel_w, pad_h, pad_w, stride_h, stride_w, hole_h, hole_w,
                tile_height, tile_width, channels, paddingc, out_height, out_width, 
                in_batch_stride, in_height_stride, in_width_stride, elems, (float*)output, fuse_param);
            } else if(type == ppl::common::DATATYPE_INT8) {
                if(func_vec[id].algo_type == SP_DEPTHWISE_KERNEL)
                {
                    dim_block.x = 128;
                    dim_grid.x  = DivUp(DivUp(out_height,4) * out_width * DivUp(channels, 4), 128);
                    dim_grid.y = conv_param.in_num;
                }
                func_vec[id].kernel_ptr_int8<<<dim_grid, dim_block, 0, stream>>>((const int8_t*)input, (const int8_t*)filter, (const float*)bias, 
                padc_fast, hw_fast, width_fast,
                in_height, in_width, kernel_h, kernel_w, pad_h, pad_w, stride_h, stride_w, hole_h, hole_w,
                tile_height, tile_width, channels, paddingc, out_height, out_width, 
                in_batch_stride, in_height_stride, in_width_stride, elems, (int8_t*)output, fuse_param, pic_scale, flt_scale, out_scale);
            }
        }
        hipEventRecord(end, stream);
        hipEventSynchronize(end);
        hipEventElapsedTime(&elapsed, begin, end);
        if (elapsed < min_time) {
            kernel_id = id;
            min_time  = elapsed;
        }
    }
    return kernel_id;
}
void PPLCUDADepthwiseForwardCudaImp(
    hipStream_t& stream,
    int kernel_id,
    void* input,
    void* filter,
    void* bias,
    conv_param_t &conv_param, 
    fuse_param_t &fuse_param,
    void* output,
    ppl::common::datatype_t type,
    float pic_scale,
    float* flt_scale,
    float out_scale)
{

    GETPARAM
    if (func_vec.empty()) InitKernelList(func_vec, type);
    int tile_height, tile_width, elems;
    GenConfigure(func_vec[kernel_id], conv_param, &tile_height, &tile_width, &elems);
    DivModFast padc_fast(paddingc);
    DivModFast hw_fast(tile_height * tile_width);
    DivModFast width_fast(tile_width);
    dim3 dim_block(BLOCK_SIZE,1,1), dim_grid(DivUp(elems, BLOCK_SIZE), 1, 1);
    if(type == ppl::common::DATATYPE_FLOAT16) {
        func_vec[kernel_id].kernel_ptr_half<<<dim_grid, dim_block, 0, stream>>>((const half*)input, (const half*)filter, (const half*)bias, 
        padc_fast, hw_fast, width_fast,
        in_height, in_width, kernel_h, kernel_w, pad_h, pad_w, stride_h, stride_w, hole_h, hole_w,
        tile_height, tile_width, channels, paddingc, out_height, out_width, 
        in_batch_stride, in_height_stride, in_width_stride, elems, (half*)output, fuse_param);
    } else if(type == ppl::common::DATATYPE_FLOAT32) {
        func_vec[kernel_id].kernel_ptr_float<<<dim_grid, dim_block, 0, stream>>>((const float*)input, (const float*)filter, (const float*)bias, 
        padc_fast, hw_fast, width_fast,
        in_height, in_width, kernel_h, kernel_w, pad_h, pad_w, stride_h, stride_w, hole_h, hole_w,
        tile_height, tile_width, channels, paddingc, out_height, out_width, 
        in_batch_stride, in_height_stride, in_width_stride, elems, (float*)output, fuse_param);
    } else if(type == ppl::common::DATATYPE_INT8) {
        out_scale = 1.0f / out_scale;
        if(func_vec[kernel_id].algo_type == SP_DEPTHWISE_KERNEL)
        {   
            dim_block.x = 128;
            dim_grid.x  = DivUp(DivUp(out_height,4) * out_width * DivUp(channels, 4), 128);
            dim_grid.y =  conv_param.in_num;
        }
        func_vec[kernel_id].kernel_ptr_int8<<<dim_grid, dim_block, 0, stream>>>((const int8_t*)input, (const int8_t*)filter, (const float*)bias, 
        padc_fast, hw_fast, width_fast,
        in_height, in_width, kernel_h, kernel_w, pad_h, pad_w, stride_h, stride_w, hole_h, hole_w,
        tile_height, tile_width, channels, paddingc, out_height, out_width, 
        in_batch_stride, in_height_stride, in_width_stride, elems, (int8_t*)output, fuse_param, pic_scale, flt_scale, out_scale);
    }
}


