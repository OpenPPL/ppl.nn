#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "cudakernel/memory/pad.h"
#include "cudakernel/common/divmod_fast.h"
#include "cudakernel/common/memory_utils.h"
#include "cudakernel/math/math.h"
#include "ppl/nn/common/tensor_shape.h"
#include "ppl/common/retcode.h"
#include <hip/hip_fp16.h>

template <typename T>
__global__ void ppl_cukernel_range(
    int64_t num_elems,
    const T* start,
    const T* delta,
    T* output)
{
    int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_elems)
        return;
    output[index] = start[0] + index * delta[0];
}

template <>
__global__ void ppl_cukernel_range<half>(
    int64_t num_elems,
    const half* start,
    const half* delta,
    half* output)
{
    typedef Math<half, half, half> OpMath;
    int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_elems)
        return;
    output[index] = OpMath::add(start[0], OpMath::mul(delta[0], __ll2half_rn(index)));
}

ppl::common::RetCode PPLCUDARangeForwardImp(
    hipStream_t stream,
    const void* start,
    const void* delta,
    ppl::nn::TensorShape* output_shape,
    void* output)
{
    int block_size     = 256;
    uint64_t num_elems = output_shape->GetElementsIncludingPadding();
    int grid_size      = (num_elems + block_size - 1) / block_size;
    switch (output_shape->GetDataType()) {
        case ppl::common::DATATYPE_FLOAT32:
            ppl_cukernel_range<float><<<grid_size, block_size, 0, stream>>>(num_elems, (float*)start, (float*)delta, (float*)output);
            break;
        case ppl::common::DATATYPE_FLOAT16:
            ppl_cukernel_range<half><<<grid_size, block_size, 0, stream>>>(num_elems, (half*)start, (half*)delta, (half*)output);
            break;
        case ppl::common::DATATYPE_INT64:
            ppl_cukernel_range<int64_t><<<grid_size, block_size, 0, stream>>>(num_elems, (int64_t*)start, (int64_t*)delta, (int64_t*)output);
            break;
        default:
            return ppl::common::RC_UNSUPPORTED;
    }
    // ppl_cukernel_range<<<grid_size, block_size, 0, stream>>>(num_elems, start, delta, (T*)output);
    return ppl::common::RC_SUCCESS;
}
