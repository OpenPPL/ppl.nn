#include "hip/hip_runtime.h"
#include "cudakernel/nn/cumsum.h"
#include <hipcub/hipcub.hpp>


constexpr inline int ceil_div(int n, int m) {
  return (n + m - 1) / m;
}

template<typename scalar_t>
__global__ void tensor_kernel_scan_outer_dim(scalar_t *tgt_, const scalar_t *src_,
                                              const int num_orows, const int num_irows, const int row_size,
                                              const scalar_t init)
{
  for (int orow = blockIdx.x; orow < num_orows; orow += gridDim.x) {
    for (int irow = blockIdx.y * blockDim.x + threadIdx.x; irow < num_irows; irow += gridDim.y * blockDim.x) {
      const scalar_t *src = src_ + orow * row_size * num_irows + irow;
      scalar_t *tgt = tgt_ + orow * row_size * num_irows + irow;
      scalar_t acc = init;

      for (int col = 0; col < row_size; ++col) {
        acc = acc + *src;
        *tgt = acc;

        src += num_irows;
        tgt += num_irows; 
      }
    }
  }
}

/* Perform an inclusive scan along the innermost dimension of a tensor.
 *
 * - num_rows is the size of the flattened outer dimensions;
 * - row_size is the size of the innermost dimension;
 *
 * The outer dimensions of the tensor are considered as a single dimension, i.e. the tensor is
 * considered as having 'num_rows' rows of size 'row_size'.
 * Each thread block processes one or more sets of contiguous rows (processing multiple rows
 * per thread block is quicker than processing a single row, especially for short rows).
 */
template<typename T, int num_threads_x, int num_threads_y>
__device__ void tensor_kernel_scan_innermost_dim_impl(T* row_buf, T *tgt_, const T *src_,
                                      const int num_rows, const int row_size,
                                      T init){
  for (int block_row = blockIdx.x * blockDim.y;
       block_row < num_rows;
       block_row += blockDim.y * gridDim.x) {
    int row = block_row + threadIdx.y;
    T block_total = init;

    const T *row_src = src_ + row * row_size;
    T *row_tgt = tgt_ + row * row_size;

    // Perform scan on one block at a time, keeping track of the total value of
    // all blocks processed so far.
    for (int block_col = 0; block_col < row_size; block_col += 2 * num_threads_x) {
      // Load data into shared memory (two values per thread).
      int col1 = block_col + threadIdx.x;
      int col2 = block_col + num_threads_x + threadIdx.x;
      if (row < num_rows) {
        if (col1 < row_size) {
          row_buf[threadIdx.x] = row_src[col1];
        } else {
          row_buf[threadIdx.x] = init;
        }

        if (col2 < row_size) {
          row_buf[num_threads_x + threadIdx.x] = row_src[col2];
        } else {
          row_buf[num_threads_x + threadIdx.x] = init;
        }

        // Add the total value of all previous blocks to the first value of this block.
        if (threadIdx.x == 0) {
          row_buf[0] = row_buf[0] + block_total;
        }
      }
      __syncthreads();

      // Parallel reduction (up-sweep). 
      for (int s = num_threads_x, d = 1; s >= 1; s >>= 1, d <<= 1) {
        if (row < num_rows && threadIdx.x < s) {
          int offset = (2 * threadIdx.x + 1) * d - 1;
          row_buf[offset + d] = row_buf[offset] + row_buf[offset + d];
        }
        __syncthreads();
      }

      // Down-sweep.
      for (int s = 2, d = num_threads_x / 2; d >= 1; s <<= 1, d >>= 1) {
        if (row < num_rows && threadIdx.x < s - 1) {
          int offset = 2 * (threadIdx.x + 1) * d - 1;
          row_buf[offset + d] = row_buf[offset] + row_buf[offset + d];
        }
        __syncthreads();
      }

      // Write back to output.
      if (row < num_rows) {
        if (col1 < row_size) row_tgt[col1] = row_buf[threadIdx.x];
        if (col2 < row_size) row_tgt[col2] = row_buf[num_threads_x + threadIdx.x];
      }
      block_total = row_buf[2 * num_threads_x - 1];
      __syncthreads();
    }
  }
}

template <
    typename T,
    int num_threads_x,
    int num_threads_y>
__global__ void tensor_kernel_scan_innermost_dim(
    T* tgt_,
    const T* src_,
    int num_rows,
    int row_size,
    T init) {
  __shared__ T sbuf[num_threads_y][2 * num_threads_x];
  T* row_buf = sbuf[threadIdx.y];

  tensor_kernel_scan_innermost_dim_impl<T, num_threads_x, num_threads_y>(
      row_buf, tgt_, src_, num_rows, row_size, init);
}




ppl::common::RetCode PPLCUDACumsumForwardImp(
    hipStream_t stream,
    int axis,
    const ppl::nn::TensorShape* input_shape,
    const void* input,
    void* output)
{
    int num_elems = input_shape->GetElementsIncludingPadding();
    int num_dims = input_shape->GetDimCount();
    int row_size = input_shape->GetDim(axis);
    if(row_size == num_elems) {
      void *d_temp_storage = NULL;
      size_t temp_storage_bytes = 0;
      hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, (const float*)input, (float*)output, num_elems);
// Allocate temporary storage for inclusive prefix sum
      hipMalloc(&d_temp_storage, temp_storage_bytes);
// Run inclusive prefix sum
      hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, (const float*)input, (float*)output, num_elems);
      hipFree(d_temp_storage);
    } else if(axis == num_dims - 1) {
        int num_rows = num_elems / row_size;
        dim3 threads(16, 32);
        dim3 grid(ceil_div(num_rows, threads.y));
        tensor_kernel_scan_innermost_dim<float, 16, 32><<<grid, threads, 0, stream>>>(
        (float*)output, (const float*)input, num_rows, row_size, 0);

    } else {
        int num_orows = 1;
        int num_irows = 1;
        for(int i = 0; i < axis; i++)
            num_orows *= input_shape->GetDim(i);
        for(int i = axis + 1; i < num_dims; i++)
            num_irows *= input_shape->GetDim(i);

        dim3 threads(std::min(512, num_irows));
        dim3 grid(num_orows, ceil_div(num_irows, threads.x));

        tensor_kernel_scan_outer_dim<float><<<grid, threads, 0, stream>>>(
            (float*)output, (const float*)input,
            num_orows, num_irows, row_size, 0);

    }
    return ppl::common::RC_SUCCESS;




}