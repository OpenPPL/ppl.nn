#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "cudakernel/gemm/bgemm.h"
#include "cudakernel/gemm/gemm.h"
#include "cudakernel/math/math.h"
#include "cudakernel/common/common.h"
#include "cudakernel/common/cuda_check.h"

#include <hip/hip_fp16.h>
#include <float.h>
#include <algorithm>

#include "kernel_type.h"
#include "conv_common.h"
#include "cudakernel/nn/conv/gene_kernel.h"

#define TIMES 4

// defined in gemm.cu
extern std::vector<kernel_info_t> g_fp16_kvec;
extern bool is_g_fp16_kvec_set;

#define FAKE_CONV_PARAM              \
    int in_hw               = 1;     \
    int out_hw              = 1;     \
    int flt_hw              = 1;     \
    int splitk              = 1;     \
    int in_height           = 1;     \
    int in_width            = 1;     \
    int conv_batch          = M;     \
    int num_grp             = 1;     \
    int num_chl_per_grp     = 0;     \
    int num_chl_per_grp_pad = K_pad; \
    int flt_height          = 1;     \
    int flt_width           = 1;     \
    int num_flt_per_grp     = N;     \
    int num_flt_per_grp_pad = N_pad; \
    int out_height          = 1;     \
    int out_width           = 1;     \
    int stride_height       = 1;     \
    int stride_width        = 1;     \
    int pad_height          = 0;     \
    int pad_width           = 0;     \
    int hole_height         = 1;     \
    int hole_width          = 1;

#define GEMM_FUNC_PARAM                                               \
        input0_tmp,                                                   \
        tmp_weight,                                                   \
        final_out,                                                    \
        kLoopNum,                                                     \
        in_lut, 0,                                                    \
        flt_lut, 0,                                                   \
        in_hw, out_hw,                                                \
        flt_hw, splitk,                                               \
        in_height, in_width,                                          \
        conv_batch, num_grp,                                          \
        num_chl_per_grp, num_chl_per_grp_pad,                         \
        flt_height, flt_width,                                        \
        num_flt_per_grp, num_flt_per_grp_pad,                         \
        out_height, out_width,                                        \
        stride_height, stride_width,                                  \
        pad_height, pad_width,                                        \
        hole_height, hole_width,                                      \
        has_bias, (int4 *)bias,                                       \
        fuse_param.has_activation, clip_min,                          \
        fuse_param.has_clip, clip_max,                                \
        fuse_param.has_prelu, (const void *)fuse_param.prelu,         \
        fuse_param.has_elt, (const int4 *)fuse_param.pre_data,        \
        fuse_param.has_elt_activation, elt_clip_min,                  \
        fuse_param.has_elt_clip, elt_clip_max,                        \
        fuse_param.has_elt_prelu, (const void *)fuse_param.elt_prelu, \
        (__half)fuse_param.leaky, (__half)fuse_param.elt_leaky,       \
        fuse_param.has_concat, concat_offset_v8,                      \
        concat_stride_v8

extern void init_f1_kvec(std::vector<kernel_info_t> &g_fp16_kvec, int device_id, ppl::common::datatype_t type);

uint64_t PPLBgemmCUDAGetBufSize(
    const ppl::nn::TensorShape *input_shape,
    int transA)
{
    return 0;
}

template <typename T>
__global__ void matrix_transpose(
    T *output,
    T *input,
    const int in_height,
    const int in_width,
    const int out_height,
    const int out_width)
{
    __shared__ T shared[32][33];

    int tid = threadIdx.x;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;

    int in_h = bx * 32;
    int in_w = by * 32 + tid;
    int out_h = by * 32;
    int out_w = bx * 32 + tid;

    int in_idx = bz * in_height * in_width + in_h * in_width + in_w;
    int out_idx = bz * out_height * out_width + out_h * out_width + out_w;

    if (in_w < in_width) {
        for (int i = 0; i < 32; i++){
            if (in_h + i < in_height)
                shared[i][tid] = input[in_idx + i * in_width];
        }
    }
    __syncthreads();
    T reg_zero = 0;
    if (out_w < out_width) {
        for (int i = 0; i < 32; i++) {
            if (out_h + i < out_height) {
                output[out_idx + i * out_width] = (out_w < in_height && (out_h + i) < in_width) ?
                                                shared[tid][i] : reg_zero;
            }
        }
    }
}

ppl::common::RetCode PPLCUDABgemmModifyWeights(
    const hipStream_t &stream,
    ppl::nn::TensorShape *weight_shape,
    void *weight,
    void *tmp_weight, // if need pad transpose
    const ppl::nn::onnx::GemmParam *param)
{
    //int transB   = param->transB;
    //float alpha  = param->alpha;
    auto type    = weight_shape->GetDataType();
    int pad_size = GetPadSize(type);

    int dim_count = weight_shape->GetDimCount();
    int batch = 1;
    for (int i = 0; i < dim_count-2; i++){
        batch *= weight_shape->GetDim(i);
    }
    const int dim0 = weight_shape->GetDim(dim_count - 2); // original shape 
    const int dim1 = weight_shape->GetDim(dim_count - 1);
    const int dim0_pad = Align(dim0, pad_size);

#define TRANSWEIGHT(Type)                                                                                      \
    matrix_transpose<Type><<<grid, block, 0, stream>>>((Type *)tmp_weight, (Type *)weight, dim0, dim1, dim1, dim0_pad); \

        dim3 grid(DivUp(dim0_pad, 32), DivUp(dim1, 32), batch);
        dim3 block(32, 1, 1);
        switch (type) {
            case ppl::common::DATATYPE_FLOAT32: {
                TRANSWEIGHT(float)
                break;
            }
            case ppl::common::DATATYPE_FLOAT16: {
                TRANSWEIGHT(__half)

                break;
            }
            default:
                return ppl::common::RC_UNSUPPORTED;
        }
#undef TRANSWEIGHT

    return ppl::common::RC_SUCCESS;
}

template<typename T>
__global__ void pad_matrix(T *input, T *output, uint64_t outer, int ori_inner, int align_inner){
    uint64_t out_off = blockIdx.x * blockDim.x + threadIdx.x;
    bool in_o_range = out_off < outer * (uint64_t)align_inner;
    int inner_id = out_off % align_inner;
    uint64_t outer_id = out_off / align_inner;
    bool in_i_range = outer_id < outer && inner_id < ori_inner;
    int in_off = outer_id * ori_inner + inner_id;
    T value = in_i_range ? input[in_off] : (T)0;

    if (in_o_range)    output[out_off] = value;
}
ppl::common::RetCode PPLCUDABgemmPadInput(
    const hipStream_t &stream,
    ppl::nn::TensorShape *input_shape,
    void *input,
    void *tmp_input, // if need transpose
    const ppl::nn::onnx::GemmParam *param)
{
    auto type    = input_shape->GetDataType();
    int pad_size = GetPadSize(type);

    int dim_count0 = input_shape->GetDimCount();
    uint64_t batch = 1;
    for (int i = 0; i < dim_count0-2; i++){
        batch *= input_shape->GetDim(i);
    }
    const int dim0 = input_shape->GetDim(dim_count0 - 2); // original shape 
    const int dim1 = input_shape->GetDim(dim_count0 - 1);
    const int dim1_pad = Align(dim1, pad_size);
    dim3 block(512,1,1);
    dim3 grid(1,1,1);
    uint64_t size = batch * dim0 * dim1_pad;
    grid.x = DivUp(size, block.x);
    //grid.y = dim0 * batch;
    switch (type) {
        case ppl::common::DATATYPE_FLOAT32: {
            pad_matrix<<<grid, block, 0, stream>>>((float*)input, (float*)tmp_input,
                                                   batch*dim0, dim1, dim1_pad);
            break;
        }
        case ppl::common::DATATYPE_FLOAT16: {
            pad_matrix<<<grid, block, 0, stream>>>((__half*)input, (__half*)tmp_input,
                                                   batch*dim0, dim1, dim1_pad);
            break;
        }
        default:
                return ppl::common::RC_UNSUPPORTED;
    }
    return ppl::common::RC_SUCCESS;
}
template<typename T>
__global__ void matrix_rm_pad(T *input, T *output, uint64_t outer, int ori_inner, int align_inner){
    uint64_t out_off = blockIdx.x * blockDim.x + threadIdx.x;
    bool in_o_range = out_off < outer * ori_inner;
    int inner_id = out_off % ori_inner;
    int outer_id = out_off / ori_inner;
    int in_off = outer_id * align_inner + inner_id;

    if (in_o_range)    output[out_off] = input[in_off];
}

ppl::common::RetCode PPLCUDABgemmCvtOutput(
    const hipStream_t &stream,
    ppl::nn::TensorShape *output_shape,
    void *output,
    void *tmp_output)
{
    auto type = output_shape->GetDataType();
    int pad_size = GetPadSize(type);

    int dim_count = output_shape->GetDimCount();
    uint64_t batch = 1;
    for (int i = 0; i < dim_count-2; i++){
        batch *= output_shape->GetDim(i);
    }
    const int dim0 = output_shape->GetDim(dim_count - 2); // original shape 
    const int dim1 = output_shape->GetDim(dim_count - 1);
    const int dim1_pad = Align(dim1, pad_size);
    dim3 block(512,1,1);
    dim3 grid(1,1,1);
    uint64_t size = batch * dim0 * dim1;
    grid.x = DivUp(size, block.x);
    //grid.y = dim0 * batch;
    switch (type) {
        case ppl::common::DATATYPE_FLOAT32: {
            matrix_rm_pad<<<grid, block, 0, stream>>>((float*)tmp_output, (float*)output,
                                                   batch*dim0, dim1, dim1_pad);
            break;
        }
        case ppl::common::DATATYPE_FLOAT16: {
            matrix_rm_pad<<<grid, block, 0, stream>>>((__half*)tmp_output, (__half*)output,
                                                   batch*dim0, dim1, dim1_pad);
            break;
        }
        default:
                return ppl::common::RC_UNSUPPORTED;
    }
    return ppl::common::RC_SUCCESS;
}

#define MAX_KERNEL_SIZE (1 + 12 + 30)

__inline__ std::string ToString(int v)
{
    std::stringstream ss;
    ss << v;
    return ss.str();
}

double PPLCUDABgemmJITSelectKernel(
    int device_id,
    hipStream_t &stream,
    ppl::common::datatype_t type,
    ppl::nn::TensorShape *input_shape,
    void *input,
    ppl::nn::TensorShape *weight_shape,
    void *weight,
    void *bias,
    ppl::nn::TensorShape *output_shape,
    void *output,
    void *temp_buffer,
    conv_param_t &conv_param,
    fuse_param_t &fuse_param,
    algo_param_t &algo_param,
    uint64_t workspace)
{
    double elapsed = 0.0f;
#ifdef PPLNN_ENABLE_CUDA_JIT
    std::vector<std::string> knames;
    std::vector<algo_param_t> params;
    std::string sources = "";

    GetFp16ConvKernelNominees(device_id, type, conv_param, knames, params, sources);

    int index = 0;
    std::vector<const char *> compile_params;
    elapsed = AlgoForwardTime(device_id, stream, knames, sources, index, compile_params, device_id, true, type, (int4 *)input, (int4 *)weight, (int4 *)output, (int4 *)bias, (int4 *)temp_buffer, params, conv_param, fuse_param, workspace);

    algo_param = params[index];
#endif
    return elapsed;
}

double PPLCUDABgemmSelectKernel(
    int device_id,
    const hipStream_t &stream,
    const ppl::nn::TensorShape *input_shape,
    const void *input,
    const ppl::nn::TensorShape *weight_shape,
    void *weight,
    const ppl::nn::TensorShape *output_shape,
    void *output,
    void *temp_buffer,
    const ppl::nn::onnx::GemmParam &param,
    const fuse_param_t &fuse_param,
    algo_param_t &algo_param)
{
#if __CUDACC_VER_MAJOR__ * 1000 + __CUDACC_VER_MINOR__ * 10 >= 9020
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, device_id);

    auto type = weight_shape->GetDataType();
    if (!is_g_fp16_kvec_set)
        init_f1_kvec(g_fp16_kvec, device_id, type);

    int pad_size = GetPadSize(type);

    // FIXME use non-paded N in conv1x1 for input
    auto dim_count0 = input_shape->GetDimCount();
    auto dim_count1 = weight_shape->GetDimCount();
    if (input_shape->GetDim(dim_count0-1) != weight_shape->GetDim(dim_count1-2))
        return FLT_MAX;
    //FIXME Dim is 64 bit?
    int m_id = dim_count0 - 2;
    while(m_id && input_shape->GetDim(m_id)==1)    m_id--;
    int M = input_shape->GetDim(m_id);
    uint64_t batch = 1;
    for (int i = 0; i < m_id; i++){
        batch *= input_shape->GetDim(i);
    }
    if (dim_count1 == 2){
        M *= batch;
        batch = 1;
    }
    int K_pad = input_shape->GetDim(dim_count0-1);
    int N     = weight_shape->GetDim(dim_count1-1);
    int N_pad = Align(N, pad_size);

    int concat_offset_v8 = fuse_param.concat_offset / pad_size;
    int concat_stride_v8 = fuse_param.concat_stride / pad_size;
    int4 *final_out      = fuse_param.has_concat ? (int4 *)fuse_param.post_concat : (int4 *)output;

    // fuse configs
    __half2 clip_min     = __float2half2_rn(fuse_param.clip_min);
    __half2 clip_max     = __float2half2_rn(fuse_param.clip_max);
    __half2 elt_clip_min = __float2half2_rn(fuse_param.elt_clip_min);
    __half2 elt_clip_max = __float2half2_rn(fuse_param.elt_clip_max);
    bool has_bias        = false; // beta != 0.f;
    half *bias = NULL;
    int4 *tmp_weight = reinterpret_cast<int4*>(weight);

    float minTime = FLT_MAX;
    int best_kid  = -1;

    float elapsed;
    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);

    // transpose
    int4 *input0_tmp = (int4 *)input;

    for (unsigned int kid = 0; kid < g_fp16_kvec.size(); kid++) {
        int tile_m_per_cta = g_fp16_kvec[kid].tile_m_per_cta;
        int tile_n_per_cta = g_fp16_kvec[kid].tile_n_per_cta;
        int tile_k_per_cta = g_fp16_kvec[kid].tile_k_per_cta;

        int cta_size_in_thd = g_fp16_kvec[kid].cta_size_in_thd;
        int smem_size       = g_fp16_kvec[kid].smem_size;

        if (!g_fp16_kvec[kid].CheckSMemSizeFeasible(device_prop))
                continue;

        if (!g_fp16_kvec[kid].CheckGpuArchFeasible(device_prop))
                continue;

        g_fp16_kvec[kid].AdaptLutKernelSMemSize();

        dim3 block_size, grid_size;
        block_size.x = cta_size_in_thd;
        block_size.y = 1;
        block_size.z = 1;

        grid_size.x = DivUp(M, tile_m_per_cta);
        grid_size.y = DivUp(N_pad, tile_n_per_cta);
        grid_size.z = 1;

        hipEventRecord(begin, stream);
        for (int i = 0; i < TIMES; i++) {
            if (g_fp16_kvec[kid].ktype == CONV_2SPK_F1) {
                FAKE_CONV_PARAM
                int kLoopNum = DivUp(K_pad, tile_k_per_cta);
                lut_t in_lut, flt_lut;
                while (batch > 65535) {
                    grid_size.z = 65535;
                    batch -= 65535;
                    (g_fp16_kvec[kid].lut_kptr)<<<grid_size, block_size, smem_size, stream>>>(GEMM_FUNC_PARAM);
                    input0_tmp += (uint64_t)65535 * M * K_pad / pad_size;// int4
                    tmp_weight += (uint64_t)65535 * N * K_pad / pad_size;// void
                    final_out += (uint64_t)65535 * M * N_pad / pad_size;// int4
                }
                if (batch > 0){
                    grid_size.z = batch;
                    (g_fp16_kvec[kid].lut_kptr)<<<grid_size, block_size, smem_size, stream>>>(GEMM_FUNC_PARAM);
                }
            }
        }

        hipEventRecord(end, stream);
        hipEventSynchronize(end);
        hipEventElapsedTime(&elapsed, begin, end);

        if (elapsed < minTime) {
            best_kid = kid;
            minTime  = elapsed;
        }
    }

    hipEventDestroy(begin);
    hipEventDestroy(end);

    algo_param.kid = best_kid;
    return minTime;
#else
    return 0.0;
#endif
}

// (B, M, K_pad) * ((B,)N, K_pad) = (B, M, N_pad)
ppl::common::RetCode PPLCUDABgemmForwardImp(
    int device_id,
    const hipStream_t &stream,
    ppl::nn::cuda::CUDAModule *module,
    const ppl::nn::TensorShape *input_shape,
    const void *input,
    const ppl::nn::TensorShape *weight_shape,
    void *weight,
    const ppl::nn::TensorShape *output_shape,
    void *output,
    const ppl::nn::onnx::GemmParam &param,
    void *temp_buffer,
    fuse_param_t &fuse_param,
    const algo_param_t &algo_param)
{
#if __CUDACC_VER_MAJOR__ * 1000 + __CUDACC_VER_MINOR__ * 10 >= 9020
    auto type = weight_shape->GetDataType();
#ifndef PPLNN_ENABLE_CUDA_JIT
    if (!is_g_fp16_kvec_set)
        init_f1_kvec(g_fp16_kvec, device_id, type);
#endif
    int pad_size = GetPadSize(type);
    //int transA   = param.transA;
    //int transB   = param.transB;
    //if (!param.transB)
    //    return ppl::common::RC_UNSUPPORTED;
    //int N     = transB ? weight_shape->GetDim(0) : weight_shape->GetDim(1);
    //int K     = transB ? weight_shape->GetDim(1) : weight_shape->GetDim(0);
    //int N_pad = Align(N, pad_size);
    //int K_pad = Align(K, pad_size);
    //int M     = transA ? input_shape->GetDim(1) : input_shape->GetDim(0);

    //FIXME Dim is 64 bit?
    //int M     = input_shape->GetDim(dim_count0 - 2);
    auto dim_count0 = input_shape->GetDimCount();
    auto dim_count1 = weight_shape->GetDimCount();
    int m_id = dim_count0 - 2;
    while(m_id && input_shape->GetDim(m_id)==1)    m_id--;
    int M = input_shape->GetDim(m_id);
    uint64_t batch = 1;
    for (int i = 0; i < m_id; i++){
        batch *= input_shape->GetDim(i);
    }
    if (dim_count1 == 2){
        M *= batch;
        batch = 1;
    }
    int K     = input_shape->GetDim(dim_count0- 1);
    int K_pad = Align(K, pad_size);
    int N     = weight_shape->GetDim(dim_count1 - 1);
    int N_pad = Align(N, pad_size);


    int concat_offset_v8 = fuse_param.concat_offset / pad_size;
    int concat_stride_v8 = fuse_param.concat_stride / pad_size;
    int4 *final_out      = fuse_param.has_concat ? (int4 *)fuse_param.post_concat : (int4 *)output;

    // fuse configs
    __half2 clip_min            = __float2half2_rn(fuse_param.clip_min);
    __half2 clip_max            = __float2half2_rn(fuse_param.clip_max);
    __half2 elt_clip_min        = __float2half2_rn(fuse_param.elt_clip_min);
    __half2 elt_clip_max        = __float2half2_rn(fuse_param.elt_clip_max);
    ppl::common::RetCode status = ppl::common::RC_SUCCESS;
    half *bias = NULL;

    // kernel configs
#ifdef PPLNN_ENABLE_CUDA_JIT
    int tile_m_per_cta  = algo_param.tiles.m_cta;
    int tile_n_per_cta  = algo_param.tiles.n_cta;
    int tile_k_per_cta  = algo_param.tiles.k_cta;
    int cta_size_in_thd = algo_param.tiles.cta_size_in_thd;
#else
    int kid             = algo_param.kid;
    int tile_m_per_cta  = g_fp16_kvec[kid].tile_m_per_cta;
    int tile_n_per_cta  = g_fp16_kvec[kid].tile_n_per_cta;
    int tile_k_per_cta  = g_fp16_kvec[kid].tile_k_per_cta;
    int cta_size_in_thd = g_fp16_kvec[kid].cta_size_in_thd;
    int smem_size       = g_fp16_kvec[kid].smem_size;
#endif
    dim3 block_size, grid_size;

    block_size.x = cta_size_in_thd;
    block_size.y = 1;
    block_size.z = 1;
    grid_size.x  = DivUp(M, tile_m_per_cta);
    grid_size.y  = DivUp(N_pad, tile_n_per_cta);
    grid_size.z  = 1; // num_grp * splitk;
    int kLoopNum = DivUp(K_pad, tile_k_per_cta);
    lut_t in_lut, flt_lut;

    bool has_bias    = bias; // beta != 0.f;
    int4 *input0_tmp = (int4 *)input;

    FAKE_CONV_PARAM
#ifdef PPLNN_ENABLE_CUDA_JIT
    int in_lut_size  = 0;
    int flt_lut_size = 0;
    void *prelu      = (void *)fuse_param.prelu;
    void *pre_data   = (void *)fuse_param.pre_data;
    void *elt_prelu  = (void *)fuse_param.elt_prelu;
    half leaky       = fuse_param.leaky;
    half elt_leaky   = fuse_param.elt_leaky;

    void *args[]        = {&input0_tmp, &weight, &final_out, &kLoopNum, &in_lut, &in_lut_size, &flt_lut, &flt_lut_size, &in_hw, &out_hw, &flt_hw, &splitk, &in_height, &in_width, &conv_batch, &num_grp, &num_chl_per_grp, &num_chl_per_grp_pad, &flt_height, &flt_width, &num_flt_per_grp, &num_flt_per_grp_pad, &out_height, &out_width, &stride_height, &stride_width, &pad_height, &pad_width, &hole_height, &hole_width, &has_bias, &bias, &fuse_param.has_activation, &clip_min, &fuse_param.has_clip, &clip_max, &fuse_param.has_prelu, &prelu, &fuse_param.has_elt, &(pre_data), &fuse_param.has_elt_activation, &elt_clip_min, &fuse_param.has_elt_clip, &elt_clip_max, &fuse_param.has_elt_prelu, &(elt_prelu), &leaky, &elt_leaky, &fuse_param.has_concat, &concat_offset_v8, &concat_stride_v8};
    hipFunction_t function = module->GetKernelFunc();
    grid_size.z = batch;
    CUDA_SAFE_CALL(hipModuleLaunchKernel(function, grid_size.x, grid_size.y, grid_size.z, block_size.x, block_size.y, block_size.z, 0, stream, args, 0));
#else
    int4 *tmp_weight = reinterpret_cast<int4*>(weight);
    while (batch > 65535) {
        grid_size.z = 65535;
        batch -= 65535;
        g_fp16_kvec[kid].AdaptLutKernelSMemSize();
        (g_fp16_kvec[kid].lut_kptr)<<<grid_size, block_size, smem_size, stream>>>(GEMM_FUNC_PARAM);
        input0_tmp += (uint64_t)65535 * M * K_pad / pad_size;// int4
        tmp_weight += (uint64_t)65535 * N * K_pad / pad_size;// void
        final_out += (uint64_t)65535 * M * N_pad / pad_size;// int4
    }
    if (batch > 0){
        grid_size.z = batch;
        g_fp16_kvec[kid].AdaptLutKernelSMemSize();
        (g_fp16_kvec[kid].lut_kptr)<<<grid_size, block_size, smem_size, stream>>>(GEMM_FUNC_PARAM);
    }
#endif
    return status;
#else
    return ppl::common::RC_UNSUPPORTED;
#endif
}
