#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include <vector>
#include <hip/hip_runtime.h>
#include <assert.h>

#include <fstream>
#include <sstream>
#include <string>
#include <unordered_map>

#include "cudakernel/nn/conv/conv_fp16.h"
#include "kernel_type.h"
#include "conv_common.h"
#include "common/init_lut.h"
#include "common/merge_split.h"

#include "float.h"

#define TIMES 4

#define SPK_KPARAM_LIST \
            pad_input,                                                                  \
            d_flt,                                                                      \
            conv_out,                                                                   \
            kloop_num,                                                                  \
    		in_lut,                        in_lut_size,                                 \
    		flt_lut,                       flt_lut_size,                                \
            chl_lut,                       chl_lut_size,                                \
            kloop_lut,                     kloop_lut_size,                              \
            in_hw,                         out_hw,                                      \
            flt_hw,                        splitk,                                      \
            conv_param.in_height,          conv_param.in_width,                         \
            conv_param.in_num,             conv_param.num_grp,                          \
            num_chl_per_grp,               num_chl_per_grp_pad,                         \
            conv_param.flt_height,         conv_param.flt_width,                        \
            num_flt_per_grp,               num_flt_per_grp_pad,                         \
            conv_param.out_height,         conv_param.out_width,                        \
            conv_param.stride_height,      conv_param.stride_width,                     \
            conv_param.pad_height,         conv_param.pad_width,                        \
            conv_param.hole_height,        conv_param.hole_width,                       \
            conv_param.has_bias,           (int *)bias

#define LUT_KPARAM_LIST \
            pad_input,                                                                  \
            d_flt,                                                                      \
            conv_out,                                                                   \
            kloop_num,                                                                  \
    		in_lut,                        in_lut_size,                                 \
    		flt_lut,                       flt_lut_size,                                \
            in_hw,                         out_hw,                                      \
            flt_hw,                        splitk,                                      \
            conv_param.in_height,          conv_param.in_width,                         \
            conv_param.in_num,             conv_param.num_grp,                          \
            num_chl_per_grp,               num_chl_per_grp_pad,                         \
            conv_param.flt_height,         conv_param.flt_width,                        \
            num_flt_per_grp,               num_flt_per_grp_pad,                         \
            conv_param.out_height,         conv_param.out_width,                        \
            conv_param.stride_height,      conv_param.stride_width,                     \
            conv_param.pad_height,         conv_param.pad_width,                        \
            conv_param.hole_height,        conv_param.hole_width,                       \
            conv_param.has_bias,           bias,                                        \
            fuse_param.has_activation,     clip_min,                                    \
            fuse_param.has_clip,           clip_max,                                    \
            fuse_param.has_prelu,          (const void *) fuse_param.prelu,             \
            fuse_param.has_elt,            (const int4 *) fuse_param.pre_data,          \
            fuse_param.has_elt_activation, elt_clip_min,                                \
            fuse_param.has_elt_clip,       elt_clip_max,                                \
            fuse_param.has_elt_prelu,      (const void *) fuse_param.elt_prelu,         \
            leaky,                         elt_leaky,                                   \
            fuse_param.has_concat,         concat_offset_v8,                            \
            concat_stride_v8


#define IDX_KPARAM_LIST \
            pad_input,                                                                  \
            d_flt,                                                                      \
            conv_out,                                                                   \
            kloop_num,                      koff_num_pad,                               \
            in_hw,                         out_hw,                                      \
            flt_hw,                        out_nhw,                                     \
            conv_param.in_height,          conv_param.in_width,                         \
            conv_param.in_num,             conv_param.num_grp,                          \
            conv_param.num_chl,            num_chl_per_grp,                             \
            in_chl_per_grp_pad,            flt_chl_per_grp_pad,                         \
            conv_param.flt_height,         conv_param.flt_width,                        \
            num_flt_per_grp,               num_flt_per_grp_pad,                         \
            conv_param.out_height,         conv_param.out_width,                        \
            conv_param.stride_height,      conv_param.stride_width,                     \
            conv_param.pad_height,         conv_param.pad_width,                        \
            conv_param.hole_height,        conv_param.hole_width,                       \
            conv_param.has_bias,           bias,                                        \
            fuse_param.has_activation,     clip_min,                                    \
            fuse_param.has_clip,           clip_max,                                    \
            fuse_param.has_prelu,          (const void *) fuse_param.prelu,             \
            fuse_param.has_elt,            (const int4 *) fuse_param.pre_data,          \
            fuse_param.has_elt_activation, elt_clip_min,                                \
            fuse_param.has_elt_clip,       elt_clip_max,                                \
            fuse_param.has_elt_prelu,      (const void *) fuse_param.elt_prelu,         \
            leaky,                         elt_leaky,                                   \
            fuse_param.has_concat,         concat_offset_v8,                            \
            concat_stride_v8

#define MERGE_KPARAM_LIST \
        	conv_out,                      final_out,                                   \
        	spk_height_v1,                 spk_width_v8,                                \
        	out_hw,                        splitk * splitf,                             \
            conv_param.has_bias,           bias,                                        \
            fuse_param.has_activation,     clip_min,                                    \
            fuse_param.has_clip,           clip_max,                                    \
            fuse_param.has_prelu,          (const void *) fuse_param.prelu,             \
            fuse_param.has_elt,            (const int4 *) fuse_param.pre_data,          \
            fuse_param.has_elt_activation, elt_clip_min,                                \
            fuse_param.has_elt_clip,       elt_clip_max,                                \
            fuse_param.has_elt_prelu,      (const void *) fuse_param.elt_prelu,         \
            leaky,                         elt_leaky,                                   \
            fuse_param.has_concat,         concat_offset_v8,                            \
            concat_stride_v8

static std::vector<kernel_info_t> g_kernel_container;
static bool is_g_kernel_container_initialized = false;

static std::unordered_map<size_t, algo_param_t> g_conv_shape_hash;

void InitializeKernelContainer(std::vector<kernel_info_t> &g_kernel_container, ppl::common::datatype_t type)
{
    if( type == ppl::common::DATATYPE_FLOAT16 ) {
        Initialize2spkConvF1KernelContainer(g_kernel_container);
        Initialize2spkConvF3KernelContainer(g_kernel_container);
        Initialize2spkConvFNKernelContainer(g_kernel_container);
        Initialize2spkConvFSKernelContainer(g_kernel_container);
                      
        InitializeIdxnConvKernelContainer(g_kernel_container);
    }
    
    is_g_kernel_container_initialized = true;
}

__inline__ std::string GetConvShapeString(conv_param_t &conv_param)
{
    return std::string("b" + std::to_string(conv_param.in_num)  + \
                       "_c" + std::to_string(conv_param.num_chl) + \
                       "_d" + std::to_string(conv_param.num_flt) + \
                       "_g" + std::to_string(conv_param.num_grp) + \
                       "_h" + std::to_string(conv_param.in_height) + \
                       "_w" + std::to_string(conv_param.in_width) + \
                       "_r" + std::to_string(conv_param.flt_height) + \
                       "_s" + std::to_string(conv_param.flt_width) + \
                       "_p" + std::to_string(conv_param.pad_height) + \
                       "_q" + std::to_string(conv_param.pad_width) + \
                       "_u" + std::to_string(conv_param.stride_height) + \
                       "_v" + std::to_string(conv_param.stride_width) + \
                       "_y" + std::to_string(conv_param.hole_height) + \
                       "_x" + std::to_string(conv_param.hole_width) + \
                       "_");
}

__inline__ size_t GetConvShapeHashKey( conv_param_t &conv_param )
{
    return std::hash<std::string>{} (GetConvShapeString(conv_param));
}

uint64_t PPLCUDAConvolutionGetCompilationBufSize(ppl::common::datatype_t type, conv_param_t &conv_param, uint64_t workspace)
{
    int pad_size = GetPadSize(type);

    uint32_t num_chl_per_grp = conv_param.num_chl / conv_param.num_grp;
    uint32_t num_flt_per_grp = conv_param.num_flt / conv_param.num_grp;

    uint32_t num_chl_per_grp_pad = Align(num_chl_per_grp, pad_size);
    uint32_t num_flt_per_grp_pad = Align(num_flt_per_grp, pad_size); 

    bool  is_in_grp_pad = num_chl_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;
    bool is_out_grp_pad = num_flt_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;

    uint32_t cvt_input_size = 0;
    uint32_t cvt_output_size = 0;

    if(is_in_grp_pad)
        cvt_input_size = GetCvtInputSize( type, conv_param, num_chl_per_grp_pad);

    if(is_out_grp_pad)
        cvt_output_size = getCvtOutputSize(type, conv_param, num_flt_per_grp_pad);

    uint32_t split_size = GetMaxSplitSize(type, conv_param, num_flt_per_grp_pad);

    uint64_t total_size = cvt_input_size + cvt_output_size + split_size;

    return total_size <= workspace ? total_size : workspace;
}
uint64_t PPLCUDAConvolutionGetRuntimeBufSize(
        ppl::common::datatype_t type,
        conv_param_t &conv_param,
        unsigned int splitk,
        unsigned int splitf,
        uint64_t workspace)
{
    int pad_size = GetPadSize(type);

    uint32_t num_chl_per_grp = conv_param.num_chl / conv_param.num_grp;
    uint32_t num_flt_per_grp = conv_param.num_flt / conv_param.num_grp;

    uint32_t num_chl_per_grp_pad = Align(num_chl_per_grp, pad_size);
    uint32_t num_flt_per_grp_pad = Align(num_flt_per_grp, pad_size); 

    bool  is_in_grp_pad = num_chl_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;
    bool is_out_grp_pad = num_flt_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;

    uint32_t cvt_input_size = 0;
    uint32_t cvt_output_size = 0;

    if(is_in_grp_pad)
        cvt_input_size = GetCvtInputSize(type, conv_param, num_chl_per_grp_pad);
    if(is_out_grp_pad)
        cvt_output_size = getCvtOutputSize(type, conv_param, num_flt_per_grp_pad);

    uint32_t split_size = 0;
    
    if(splitk > 1 || splitf > 1)
        split_size = GetSplitKFSize(type, conv_param, num_flt_per_grp_pad, splitk, splitf);

    uint64_t total_size  = cvt_input_size + cvt_output_size + split_size;

    return total_size <= workspace ? total_size : workspace;
}

ppl::common::RetCode PPLCUDAConvolutionSelectKernel(
        hipStream_t &stream, 
        ppl::common::datatype_t type,
        int4* d_input,
        int4* d_flt,
        int4* d_output,
        int4* bias,
        int4* d_temp_buf, 
        algo_param_t & algo_param,
        conv_param_t &conv_param, 
        fuse_param_t &fuse_param,
	    uint64_t workspace)
{
    if(!is_g_kernel_container_initialized)
        InitializeKernelContainer(g_kernel_container, type);

    size_t conv_shape_hash = GetConvShapeHashKey(conv_param);

    std::unordered_map<size_t, algo_param_t>::const_iterator conv_shape_hash_iterator = g_conv_shape_hash.find(conv_shape_hash);

    if(conv_shape_hash_iterator != g_conv_shape_hash.end()) {
        algo_param.kid    = conv_shape_hash_iterator->second.kid;
        algo_param.splitk = conv_shape_hash_iterator->second.splitk;
        algo_param.splitf = conv_shape_hash_iterator->second.splitf;

        return ppl::common::RC_SUCCESS;
    }

    int pad_size = GetPadSize(type);

    int num_chl_per_grp = conv_param.num_chl / conv_param.num_grp;
    int num_flt_per_grp = conv_param.num_flt / conv_param.num_grp;

    int num_chl_per_grp_pad = Align(num_chl_per_grp, pad_size);
    int num_flt_per_grp_pad = Align(num_flt_per_grp, pad_size);

    int in_hw = conv_param.in_height * conv_param.in_width;
    int flt_hw = conv_param.flt_height * conv_param.flt_width;
    int out_hw = conv_param.out_height * conv_param.out_width;

    int concat_offset_v8 = fuse_param.concat_offset / pad_size;
    int concat_stride_v8 = fuse_param.concat_stride / pad_size;

    bool  is_in_grp_pad = num_chl_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;
    bool is_out_grp_pad = num_flt_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;

    uint64_t buf_off_v4 = 0;

    int4 *pad_input = d_input;
    int4 *pad_output = d_output;

    if(is_in_grp_pad) {
	    pad_input = d_temp_buf; 
	    buf_off_v4 += GetCvtInputSize(type, conv_param, num_chl_per_grp_pad) / (_4INT_TO_INT4_ * _INT_TO_4BYTE_);

        PPLCUDAConvolutionCvtInput(stream, pad_input, d_input, type, conv_param);
    }

    if(is_out_grp_pad) {
	    pad_output = d_temp_buf + buf_off_v4;
	    buf_off_v4 += getCvtOutputSize(type, conv_param, num_flt_per_grp_pad) / (_4INT_TO_INT4_ * _INT_TO_4BYTE_);
    } 

    int4 * final_out = fuse_param.has_concat ? (int4 *) fuse_param.post_concat : pad_output;

    int4 *splitk_buf = d_temp_buf + buf_off_v4;

    __half2 clip_min     = __float2half2_rn(fuse_param.clip_min);
    __half2 clip_max     = __float2half2_rn(fuse_param.clip_max);
    __half2 elt_clip_min = __float2half2_rn(fuse_param.elt_clip_min);
    __half2 elt_clip_max = __float2half2_rn(fuse_param.elt_clip_max);
    __half  leaky        = __float2half(fuse_param.leaky);
    __half  elt_leaky    = __float2half(fuse_param.elt_leaky);

    float minTime = FLT_MAX;

    float elapsed;
    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);

    const int SPLITK_OPTIONS[] = {1, 2, 4, 8};

    for(unsigned int spk = 0; spk < 4; spk++) {
        unsigned int splitk = SPLITK_OPTIONS[spk];

        for(unsigned int kid = 0; kid < g_kernel_container.size(); kid++) {
            unsigned int splitf = (g_kernel_container[kid].ktype == CONV_2SPK_FS) ? flt_hw : 1;
        
            if(!g_kernel_container[kid].CheckKernelTypeFeasible(conv_param.flt_height, conv_param.flt_width, num_chl_per_grp, splitk)) continue;

            if(!g_kernel_container[kid].CheckSplitkFeasible(num_chl_per_grp, splitk)) continue;

            if(!g_kernel_container[kid].CheckSplitfFeasible(splitf, splitk)) continue;

            int4 *conv_out = (splitk > 1 || splitf > 1) ? splitk_buf : final_out;

            dim3 block_size, grid_size;

            block_size.x = g_kernel_container[kid].cta_size_in_thd;
            block_size.y = 1;
            block_size.z = 1;

            grid_size.x = DivUp(conv_param.in_num * conv_param.out_height * conv_param.out_width, g_kernel_container[kid].tile_m_per_cta);
            grid_size.y = DivUp(num_flt_per_grp_pad, g_kernel_container[kid].tile_n_per_cta);
            grid_size.z = conv_param.num_grp * splitk * splitf;

	        hipEventRecord(begin, stream);

	        for(int i = 0; i < TIMES; i++) {
                if(g_kernel_container[kid].ktype == CONV_IDXN_C2 || g_kernel_container[kid].ktype == CONV_IDXN_C4 || \
                        g_kernel_container[kid].ktype == CONV_IDXN_C32) {
                    int tile_k_per_step = g_kernel_container[kid].tile_k_per_step;

                    int img_pad_size    = pad_size;
                    int flt_pad_size    = g_kernel_container[kid].flt_pad_size;
                    int out_nhw         = out_hw * conv_param.in_num;

                    int in_chl_per_grp_pad = Align(num_chl_per_grp, img_pad_size);
                    int flt_chl_per_grp_pad = Align(num_chl_per_grp, flt_pad_size);
                    int num_flt_per_grp_pad = Align(num_flt_per_grp, img_pad_size);

	                int kloop_num        = DivUp(flt_hw * flt_chl_per_grp_pad, g_kernel_container[kid].tile_k_per_cta);
                    int koff_num_pad      = Align(kloop_num * (g_kernel_container[kid].tile_k_per_cta / flt_pad_size), WARP_SIZE);

                    (g_kernel_container[kid].idx_kptr)<<<grid_size, block_size, 0, stream>>>(IDX_KPARAM_LIST);
                }
                else if(g_kernel_container[kid].ktype == CONV_2SPK_F1 || g_kernel_container[kid].ktype == CONV_2SPK_F3 || \
                        g_kernel_container[kid].ktype == CONV_2SPK_FN || g_kernel_container[kid].ktype == CONV_2SPK_FS) {

	                int kloop_num = (flt_hw / splitf) * DivUp(num_chl_per_grp_pad, g_kernel_container[kid].tile_k_per_cta);

                    lut_t in_lut, flt_lut;
                    int in_lut_size, flt_lut_size;
                
                    InitializeInputLut(in_lut_size, in_lut.idx, conv_param.flt_height, conv_param.flt_width, conv_param.in_height,
                            conv_param.in_width, conv_param.pad_height, conv_param.pad_width, conv_param.hole_height, conv_param.hole_width,
                            num_chl_per_grp_pad, conv_param.num_grp, g_kernel_container[kid].tile_k_per_cta, pad_size);

                    InitializeFilterLut(flt_lut_size, flt_lut.idx, conv_param.flt_height, conv_param.flt_width, num_chl_per_grp_pad,
                            g_kernel_container[kid].tile_k_per_cta, pad_size);

                    if(splitk == 1) {
                        (g_kernel_container[kid].lut_kptr)<<<grid_size, block_size, 0, stream>>>(LUT_KPARAM_LIST);
                    } else {
                        int chl_lut_size, kloop_lut_size;
                        struct chl_lut_t chl_lut;
                        struct kloop_lut_t kloop_lut;

                        InitializeChlLut(chl_lut_size, chl_lut.idx, conv_param.num_chl, conv_param.num_grp, pad_size,
                                g_kernel_container[kid].tile_k_per_cta, splitk);
                        InitializeKloopLut(kloop_lut_size, kloop_lut.idx, conv_param.num_chl, conv_param.num_grp, pad_size,
                                g_kernel_container[kid].tile_k_per_cta, splitk, splitf, flt_hw);
    
                        (g_kernel_container[kid].spk_kptr)<<<grid_size, block_size, 0, stream>>>(SPK_KPARAM_LIST);
                    }

                    if(splitk > 1 || splitf > 1) {
                        int spk_width_v8   = num_flt_per_grp_pad * conv_param.num_grp / pad_size;
                        int spk_height_v1  = out_hw * conv_param.in_num;

                        dim3 merge_grid_size, merge_block_size;
                        merge_block_size.x = 64; // empirical value
                        merge_block_size.y = 1;
                        merge_block_size.z = 1;

                        merge_grid_size.x  = spk_height_v1;
                        merge_grid_size.y  = DivUp(spk_width_v8, merge_block_size.x);
                        merge_grid_size.z  = 1;

                        MergeConvSplitResults<<<merge_grid_size, merge_block_size, 0, stream>>>(MERGE_KPARAM_LIST);
                    }
                }
            }

	        hipEventRecord(end, stream);
	        hipEventSynchronize(end);
	        hipEventElapsedTime(&elapsed, begin, end);

	        if(elapsed < minTime){
                algo_param.kid = kid;
                algo_param.splitk = splitk;
                algo_param.splitf = splitf;
	            minTime = elapsed;
	        }
        }
    }

    if(is_out_grp_pad) {
        PPLCUDAConvolutionCvtOutput(stream, d_output, final_out, type, conv_param);
    }

    hipEventDestroy(begin);
    hipEventDestroy(end);

    g_conv_shape_hash[conv_shape_hash] = algo_param;

    return ppl::common::RC_SUCCESS;
}

void PPLCUDAConvolutionForwardImp(
        hipStream_t &stream, 
        ppl::common::datatype_t type,
        int4* d_input,
        int4* d_flt,
        int4* d_output,
        int4* bias,
        int4* d_temp_buf,
        algo_param_t& algo_param,
        conv_param_t &conv_param,
        fuse_param_t &fuse_param)
{
    if(!is_g_kernel_container_initialized)
        InitializeKernelContainer(g_kernel_container, type);

    unsigned int kid = algo_param.kid;
    unsigned int splitk = algo_param.splitk;
    unsigned int splitf = algo_param.splitf;

    int pad_size = GetPadSize(type);

    int num_chl_per_grp = conv_param.num_chl / conv_param.num_grp;
    int num_flt_per_grp = conv_param.num_flt / conv_param.num_grp;

    int num_chl_per_grp_pad = Align(num_chl_per_grp, pad_size);
    int num_flt_per_grp_pad = Align(num_flt_per_grp, pad_size);

    int in_hw  = conv_param.in_height * conv_param.in_width;
    int flt_hw = conv_param.flt_height * conv_param.flt_width;
    int out_hw = conv_param.out_height * conv_param.out_width;

    int concat_offset_v8 = fuse_param.concat_offset / pad_size;
    int concat_stride_v8 = fuse_param.concat_stride / pad_size;

    bool  is_in_grp_pad = num_chl_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;
    bool is_out_grp_pad = num_flt_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;

    uint64_t buf_off_v4 = 0;

    int4 *pad_input = d_input;
    int4 *pad_output = d_output;

    if(is_in_grp_pad) {
	    pad_input = d_temp_buf; 
	    buf_off_v4 += GetCvtInputSize(type, conv_param, num_chl_per_grp_pad) / (_4INT_TO_INT4_ * _INT_TO_4BYTE_);

        PPLCUDAConvolutionCvtInput(stream, pad_input, d_input, type, conv_param);
    }

    if(is_out_grp_pad) {
	    pad_output = d_temp_buf + buf_off_v4;
	    buf_off_v4 += getCvtOutputSize(type, conv_param, num_flt_per_grp_pad) / (_4INT_TO_INT4_ * _INT_TO_4BYTE_);
    } 

    int4 *final_out  = fuse_param.has_concat ? (int4 *) fuse_param.post_concat : pad_output;

    int4 *splitk_buf = d_temp_buf + buf_off_v4;
    int4 *conv_out   = (splitk > 1 || splitf > 1) ? splitk_buf : final_out;

    __half2 clip_min     = __float2half2_rn(fuse_param.clip_min);
    __half2 clip_max     = __float2half2_rn(fuse_param.clip_max);
    __half2 elt_clip_min = __float2half2_rn(fuse_param.elt_clip_min);
    __half2 elt_clip_max = __float2half2_rn(fuse_param.elt_clip_max);
    __half  leaky        = __float2half(fuse_param.leaky);
    __half  elt_leaky    = __float2half(fuse_param.elt_leaky);

    dim3 block_size, grid_size;

    block_size.x = g_kernel_container[kid].cta_size_in_thd;
    block_size.y = 1;
    block_size.z = 1;

    grid_size.x  = DivUp(conv_param.in_num * conv_param.out_height * conv_param.out_width, g_kernel_container[kid].tile_m_per_cta);
    grid_size.y  = DivUp(num_flt_per_grp_pad, g_kernel_container[kid].tile_n_per_cta);
    grid_size.z  = conv_param.num_grp * splitk * splitf;

    if(g_kernel_container[kid].ktype == CONV_IDXN_C2 || g_kernel_container[kid].ktype == CONV_IDXN_C4 || \
            g_kernel_container[kid].ktype == CONV_IDXN_C32) {
        int img_pad_size = pad_size;
        int flt_pad_size = g_kernel_container[kid].flt_pad_size;

        int out_nhw = out_hw * conv_param.in_num;

        int in_chl_per_grp_pad = Align(num_chl_per_grp, img_pad_size);
        int flt_chl_per_grp_pad = Align(num_chl_per_grp, flt_pad_size);
        int num_flt_per_grp_pad = Align(num_flt_per_grp, img_pad_size);

	    int kloop_num = DivUp(flt_hw * flt_chl_per_grp_pad, g_kernel_container[kid].tile_k_per_cta);
        int koff_num_pad = Align(kloop_num * (g_kernel_container[kid].tile_k_per_cta / flt_pad_size), WARP_SIZE);

        (g_kernel_container[kid].idx_kptr)<<<grid_size, block_size, 0, stream>>>(IDX_KPARAM_LIST);

    } else if(g_kernel_container[kid].ktype == CONV_2SPK_F1 || g_kernel_container[kid].ktype == CONV_2SPK_F3 || \
            g_kernel_container[kid].ktype == CONV_2SPK_FN || g_kernel_container[kid].ktype == CONV_2SPK_FS ) {

	    int kloop_num = (flt_hw / splitf) * DivUp(num_chl_per_grp_pad, g_kernel_container[kid].tile_k_per_cta);

        lut_t in_lut, flt_lut;
        int in_lut_size, flt_lut_size;
    
        InitializeInputLut(in_lut_size, in_lut.idx, conv_param.flt_height, conv_param.flt_width, conv_param.in_height,
                conv_param.in_width, conv_param.pad_height, conv_param.pad_width, conv_param.hole_height, conv_param.hole_width,
                num_chl_per_grp_pad, conv_param.num_grp, g_kernel_container[kid].tile_k_per_cta, pad_size);

        InitializeFilterLut(flt_lut_size, flt_lut.idx, conv_param.flt_height, conv_param.flt_width, num_chl_per_grp_pad,
                g_kernel_container[kid].tile_k_per_cta, pad_size);

        if(splitk == 1) {
            (g_kernel_container[kid].lut_kptr)<<<grid_size, block_size, 0, stream>>>(LUT_KPARAM_LIST);
        } else {
            int chl_lut_size, kloop_lut_size;
            struct chl_lut_t chl_lut;
            struct kloop_lut_t kloop_lut;

            InitializeChlLut(chl_lut_size, chl_lut.idx, conv_param.num_chl, conv_param.num_grp, pad_size,
                    g_kernel_container[kid].tile_k_per_cta, splitk);
            InitializeKloopLut(kloop_lut_size, kloop_lut.idx, conv_param.num_chl, conv_param.num_grp, pad_size,
                    g_kernel_container[kid].tile_k_per_cta, splitk, splitf, flt_hw);

            (g_kernel_container[kid].spk_kptr)<<<grid_size, block_size, 0, stream>>>(SPK_KPARAM_LIST);
        }
    }
    
    if(splitk > 1 || splitf > 1) {
        int spk_width_v8   = num_flt_per_grp_pad * conv_param.num_grp / pad_size;
        int spk_height_v1  = out_hw * conv_param.in_num;

        dim3 merge_grid_size, merge_block_size;
        merge_block_size.x = 64;
        merge_block_size.y = 1;
        merge_block_size.z = 1;

        merge_grid_size.x  = spk_height_v1;
        merge_grid_size.y  = DivUp(spk_width_v8, merge_block_size.x);
        merge_grid_size.z  = 1;

        MergeConvSplitResults<<<merge_grid_size, merge_block_size, 0, stream>>>(MERGE_KPARAM_LIST);
    }

    if(is_out_grp_pad) {
        PPLCUDAConvolutionCvtOutput(stream, d_output, final_out, type, conv_param);
    }
    
}
